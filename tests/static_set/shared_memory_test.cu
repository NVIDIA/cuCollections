#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <test_utils.hpp>

#include <cuco/static_set.cuh>

#include <cuda/functional>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>
#include <thrust/tuple.h>

#include <catch2/catch_template_test_macros.hpp>

#include <limits>

template <std::size_t NumWindows, typename Ref>
__global__ void shared_memory_test_kernel(Ref* sets,
                                          typename Ref::key_type const* const insterted_keys,
                                          size_t number_of_elements,
                                          bool* const keys_exist,
                                          bool* const keys_correct)
{
  // Each block processes one set
  const size_t set_id = blockIdx.x;
  const size_t offset = set_id * number_of_elements;

  __shared__ typename Ref::window_type sm_buffer[NumWindows];

  auto g          = cuco::test::cg::this_thread_block();
  auto insert_ref = sets[set_id].make_copy(g, sm_buffer, cuco::thread_scope_block);
  auto find_ref   = insert_ref.with_operators(cuco::op::find);

  for (int i = g.thread_rank(); i < number_of_elements; i += g.size()) {
    auto found_it = find_ref.find(insterted_keys[offset + i]);

    if (found_it != find_ref.end()) {
      keys_exist[offset + i] = true;
      if (*found_it == insterted_keys[offset + i]) {
        keys_correct[offset + i] = true;
      } else {
        keys_correct[offset + i] = false;
      }
    } else {
      keys_exist[offset + i]   = false;
      keys_correct[offset + i] = true;
    }
  }
}

TEMPLATE_TEST_CASE_SIG("Shared memory static set", "", ((typename Key), Key), (int32_t), (int64_t))
{
  constexpr std::size_t number_of_sets  = 1000;
  constexpr std::size_t elements_in_set = 500;
  constexpr std::size_t set_capacity    = 2 * elements_in_set;

  using extent_type = cuco::extent<std::size_t, set_capacity>;
  using set_type    = cuco::static_set<Key,
                                    extent_type,
                                    cuda::thread_scope_device,
                                    thrust::equal_to<Key>,
                                    cuco::linear_probing<1, cuco::default_hash_function<Key>>>;

  // one array for all sets, first elements_in_set element belong to set 0, second to set 1 and so
  // on
  thrust::device_vector<Key> d_keys(number_of_sets * elements_in_set);

  thrust::sequence(thrust::device, d_keys.begin(), d_keys.end());

  // using std::unique_ptr because static_set does not have copy/move constructor/assignment
  // operator yet
  std::vector<std::unique_ptr<set_type>> sets;
  for (std::size_t set_id = 0; set_id < number_of_sets; ++set_id) {
    sets.push_back(std::make_unique<set_type>(extent_type{}, cuco::empty_key<Key>{-1}));
  }

  thrust::device_vector<bool> d_keys_exist(number_of_sets * elements_in_set);
  thrust::device_vector<bool> d_keys_correct(number_of_sets * elements_in_set);

  using ref_type = typename set_type::ref_type<cuco::op::insert_tag>;

  SECTION("Keys are all found after insertion.")
  {
    std::vector<ref_type> h_refs;
    for (std::size_t set_id = 0; set_id < number_of_sets; ++set_id) {
      const std::size_t offset = set_id * elements_in_set;

      set_type* set = sets[set_id].get();
      set->insert(d_keys.begin() + offset, d_keys.begin() + offset + elements_in_set);
      h_refs.push_back(set->ref(cuco::op::insert));
    }
    thrust::device_vector<ref_type> d_refs(h_refs);

    auto constexpr num_windows = cuco::make_window_extent<ref_type>(extent_type{});

    shared_memory_test_kernel<num_windows.value(), ref_type>
      <<<number_of_sets, 64>>>(d_refs.data().get(),
                               d_keys.data().get(),
                               elements_in_set,
                               d_keys_exist.data().get(),
                               d_keys_correct.data().get());

    REQUIRE(d_keys_exist.size() == d_keys_correct.size());
    auto zip =
      thrust::make_zip_iterator(thrust::make_tuple(d_keys_exist.begin(), d_keys_correct.begin()));

    REQUIRE(cuco::test::all_of(zip,
                               zip + d_keys_exist.size(),
                               cuda::proclaim_return_type<bool>([] __device__(auto const& z) {
                                 return thrust::get<0>(z) and thrust::get<1>(z);
                               })));
  }

  SECTION("No key is found before insertion.")
  {
    std::vector<ref_type> h_refs;
    for (std::size_t set_id = 0; set_id < number_of_sets; ++set_id) {
      h_refs.push_back(sets[set_id].get()->ref(cuco::op::insert));
    }
    thrust::device_vector<ref_type> d_refs(h_refs);

    auto constexpr num_windows = cuco::make_window_extent<ref_type>(extent_type{});

    shared_memory_test_kernel<num_windows.value(), ref_type>
      <<<number_of_sets, 64>>>(d_refs.data().get(),
                               d_keys.data().get(),
                               elements_in_set,
                               d_keys_exist.data().get(),
                               d_keys_correct.data().get());

    REQUIRE(cuco::test::none_of(d_keys_exist.begin(), d_keys_exist.end(), thrust::identity{}));
  }
}

auto constexpr cg_size     = 1;
auto constexpr window_size = 1;

template <std::size_t NumWindows>
__global__ void shared_memory_hash_set_kernel(bool* key_found)
{
  using Key       = int32_t;
  using slot_type = Key;

  __shared__ cuco::window<slot_type, window_size> set[NumWindows];

  using extent_type      = cuco::extent<std::size_t, NumWindows>;
  using storage_ref_type = cuco::aow_storage_ref<slot_type, window_size, extent_type>;

  auto raw_ref =
    cuco::static_set_ref{cuco::empty_key<Key>{-1},
                         thrust::equal_to<Key>{},
                         cuco::linear_probing<cg_size, cuco::default_hash_function<Key>>{},
                         cuco::thread_scope_block,
                         storage_ref_type{extent_type{}, set}};

  auto const block = cooperative_groups::this_thread_block();
  raw_ref.initialize(block);

  auto const index = threadIdx.x + blockIdx.x * blockDim.x;
  auto const rank  = block.thread_rank();

  // insert {thread_rank, thread_rank} for each thread in thread-block
  auto insert_ref = raw_ref.with_operators(cuco::op::insert);
  insert_ref.insert(rank);
  block.sync();

  auto find_ref           = insert_ref.with_operators(cuco::op::find);
  auto const retrieved_it = find_ref.find(rank);
  block.sync();

  if (retrieved_it != find_ref.end() && *retrieved_it == rank) { key_found[index] = true; }
}

TEST_CASE("static set shared memory slots.", "")
{
  constexpr std::size_t N = 256;
  [[maybe_unused]] auto constexpr num_windows =
    cuco::make_window_extent<cg_size, window_size>(cuco::extent<std::size_t, N>{});

  thrust::device_vector<bool> key_found(N, false);
  shared_memory_hash_set_kernel<num_windows.value()><<<8, 32>>>(key_found.data().get());
  CUCO_CUDA_TRY(hipDeviceSynchronize());

  REQUIRE(cuco::test::all_of(key_found.begin(), key_found.end(), thrust::identity<bool>{}));
}
