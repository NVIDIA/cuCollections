/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuco/static_set.cuh>

#include <catch2/catch_test_macros.hpp>

TEST_CASE("Static set capacity", "")
{
  using Key        = int32_t;
  using ProbeT     = cuco::double_hashing<1, cuco::default_hash_function<Key>>;
  using Equal      = thrust::equal_to<Key>;
  using AllocatorT = cuco::cuda_allocator<cuda::std::byte>;
  using StorageT   = cuco::storage<2>;

  SECTION("zero capacity is allowed.")
  {
    auto constexpr gold_capacity = 4;

    using extent_type = cuco::extent<std::size_t, 0>;
    cuco::
      static_set<Key, extent_type, cuda::thread_scope_device, Equal, ProbeT, AllocatorT, StorageT>
        set{extent_type{}, cuco::empty_key<Key>{-1}};
    auto const capacity = set.capacity();
    REQUIRE(capacity == gold_capacity);

    auto ref                = set.ref(cuco::insert);
    auto const ref_capacity = ref.capacity();
    REQUIRE(ref_capacity == gold_capacity);
  }

  SECTION("negative capacity (ikr -_-||) is also allowed.")
  {
    auto constexpr gold_capacity = 4;

    using extent_type = cuco::extent<int32_t>;
    cuco::
      static_set<Key, extent_type, cuda::thread_scope_device, Equal, ProbeT, AllocatorT, StorageT>
        set{extent_type{-10}, cuco::empty_key<Key>{-1}};
    auto const capacity = set.capacity();
    REQUIRE(capacity == gold_capacity);

    auto ref                = set.ref(cuco::insert);
    auto const ref_capacity = ref.capacity();
    REQUIRE(ref_capacity == gold_capacity);
  }

  constexpr std::size_t num_keys{400};

  SECTION("Static window extent can be evaluated at build time.")
  {
    std::size_t constexpr gold_extent = 211;

    using extent_type = cuco::extent<std::size_t, num_keys>;
    cuco::
      static_set<Key, extent_type, cuda::thread_scope_device, Equal, ProbeT, AllocatorT, StorageT>
        set{extent_type{}, cuco::empty_key<Key>{-1}};

    auto ref               = set.ref(cuco::insert);
    auto const num_windows = ref.window_extent();
    STATIC_REQUIRE(static_cast<std::size_t>(num_windows) == gold_extent);
  }

  SECTION("Dynamic extent is evaluated at run time.")
  {
    auto constexpr gold_capacity = 422;  // 211 x 2

    using extent_type = cuco::extent<std::size_t>;
    cuco::
      static_set<Key, extent_type, cuda::thread_scope_device, Equal, ProbeT, AllocatorT, StorageT>
        set{num_keys, cuco::empty_key<Key>{-1}};
    auto const capacity = set.capacity();
    REQUIRE(capacity == gold_capacity);

    auto ref                = set.ref(cuco::insert);
    auto const ref_capacity = ref.capacity();
    REQUIRE(ref_capacity == gold_capacity);
  }

  SECTION("Set can be constructed from plain integer.")
  {
    auto constexpr gold_capacity = 422;  // 211 x 2

    cuco::
      static_set<Key, std::size_t, cuda::thread_scope_device, Equal, ProbeT, AllocatorT, StorageT>
        set{num_keys, cuco::empty_key<Key>{-1}};
    auto const capacity = set.capacity();
    REQUIRE(capacity == gold_capacity);

    auto ref                = set.ref(cuco::insert);
    auto const ref_capacity = ref.capacity();
    REQUIRE(ref_capacity == gold_capacity);
  }

  SECTION("Set can be constructed from plain integer and load factor.")
  {
    auto constexpr gold_capacity = 502;  // 251 x 2

    cuco::
      static_set<Key, std::size_t, cuda::thread_scope_device, Equal, ProbeT, AllocatorT, StorageT>
        set{num_keys, 0.8, cuco::empty_key<Key>{-1}};
    auto const capacity = set.capacity();
    REQUIRE(capacity == gold_capacity);

    auto ref                = set.ref(cuco::insert);
    auto const ref_capacity = ref.capacity();
    REQUIRE(ref_capacity == gold_capacity);
  }

  SECTION("Dynamic extent is evaluated at run time.")
  {
    auto constexpr gold_capacity = 412;  // 103 x 2 x 2

    using probe = cuco::linear_probing<2, cuco::default_hash_function<Key>>;
    auto set    = cuco::static_set<Key,
                                   cuco::extent<std::size_t>,
                                   cuda::thread_scope_device,
                                   Equal,
                                   probe,
                                   AllocatorT,
                                   StorageT>{num_keys, cuco::empty_key<Key>{-1}};

    auto const capacity = set.capacity();
    REQUIRE(capacity == gold_capacity);

    auto ref                = set.ref(cuco::insert);
    auto const ref_capacity = ref.capacity();
    REQUIRE(ref_capacity == gold_capacity);
  }
}
