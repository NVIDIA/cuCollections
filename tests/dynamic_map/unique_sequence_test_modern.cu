/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <test_utils.hpp>

#include <cuco/dynamic_map.cuh>

#include <cuda/functional>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>
#include <thrust/tuple.h>

#include <catch2/catch_template_test_macros.hpp>

TEMPLATE_TEST_CASE_SIG("Unique sequence of keys modern",
                       "",
                       ((typename Key, typename T), Key, T),
                       (int32_t, int32_t),
                       (int32_t, int64_t),
                       (int64_t, int32_t),
                       (int64_t, int64_t))
{
  constexpr std::size_t num_keys{50'000'000};

  cuco::modern::dynamic_map<Key, T> map{
    30'000'000, cuco::empty_key<Key>{-1}, cuco::empty_value<T>{-1}};

  thrust::device_vector<Key> d_keys(num_keys);
  thrust::device_vector<T> d_values(num_keys);

  thrust::sequence(thrust::device, d_keys.begin(), d_keys.end());
  thrust::sequence(thrust::device, d_values.begin(), d_values.end());

  auto pairs_begin =
    thrust::make_transform_iterator(thrust::make_counting_iterator<int>(0),
                                    cuda::proclaim_return_type<cuco::pair<Key, T>>(
                                      [] __device__(auto i) { return cuco::pair<Key, T>(i, i); }));

  thrust::device_vector<T> d_results(num_keys);
  thrust::device_vector<bool> d_contained(num_keys);

  // bulk function test cases

  SECTION("All inserted keys-value pairs should be contained")
  {
    map.insert(pairs_begin, pairs_begin + num_keys);
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(cuco::test::all_of(d_contained.begin(), d_contained.end(), thrust::identity{}));
  }

  SECTION("Non-inserted keys-value pairs should not be contained")
  {
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(cuco::test::none_of(d_contained.begin(), d_contained.end(), thrust::identity{}));
  }
}
