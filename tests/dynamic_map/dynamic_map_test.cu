#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <algorithm>
#include <catch2/catch.hpp>
#include <cuco/dynamic_map.cuh>
#include <random>
#include <util.hpp>

enum class dist_type { UNIQUE, UNIFORM, GAUSSIAN };

template <dist_type Dist, typename Key, typename OutputIt>
static void generate_keys(OutputIt output_begin, OutputIt output_end)
{
  auto num_keys = std::distance(output_begin, output_end);

  std::random_device rd;
  std::mt19937 gen{rd()};

  switch (Dist) {
    case dist_type::UNIQUE:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = i;
      }
      break;
    case dist_type::UNIFORM:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(gen()));
      }
      break;
    case dist_type::GAUSSIAN:
      std::normal_distribution<> dg{1e9, 1e7};
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(dg(gen)));
      }
      break;
  }
}

TEMPLATE_TEST_CASE_SIG("Unique sequence of keys",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int64_t, dist_type::UNIQUE),
                       (int32_t, dist_type::UNIFORM),
                       (int64_t, dist_type::UNIFORM),
                       (int32_t, dist_type::GAUSSIAN),
                       (int64_t, dist_type::GAUSSIAN))
{
  using Key   = T;
  using Value = T;

  constexpr std::size_t num_keys{50'000'000};
  cuco::dynamic_map<Key, Value> map{30'000'000, -1, -1};

  std::vector<Key> h_keys(num_keys);
  std::vector<Value> h_values(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_keys<Dist, Key>(h_keys.begin(), h_keys.end());

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_values[i]       = val;
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
  }

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<Value> d_values(h_values);
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);
  thrust::device_vector<Value> d_results(num_keys);
  thrust::device_vector<bool> d_contained(num_keys);

  // bulk function test cases
  SECTION("All inserted keys-value pairs should be correctly recovered during find")
  {
    map.insert(d_pairs.begin(), d_pairs.end());
    map.find(d_keys.begin(), d_keys.end(), d_results.begin());
    auto zip = thrust::make_zip_iterator(thrust::make_tuple(d_results.begin(), d_values.begin()));

    REQUIRE(all_of(zip, zip + num_keys, [] __device__(auto const& p) {
      return thrust::get<0>(p) == thrust::get<1>(p);
    }));
  }

  SECTION("All non-inserted keys-value pairs should have the empty sentinel value recovered")
  {
    map.find(d_keys.begin(), d_keys.end(), d_results.begin());

    REQUIRE(
      all_of(d_results.begin(), d_results.end(), [] __device__(auto const& p) { return p == -1; }));
  }

  SECTION("All inserted keys-value pairs should be contained")
  {
    map.insert(d_pairs.begin(), d_pairs.end());
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(
      all_of(d_contained.begin(), d_contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("Non-inserted keys-value pairs should not be contained")
  {
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(
      none_of(d_contained.begin(), d_contained.end(), [] __device__(bool const& b) { return b; }));
  }
}