/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utils.hpp>

#include <cuco/static_map.cuh>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/sort.h>

#include <catch2/catch_template_test_macros.hpp>

#include <cuda/functional>

using size_type = std::size_t;

template <typename Map>
void test_insert_or_assign(Map& map, size_type num_keys)
{
  using Key   = typename Map::key_type;
  using Value = typename Map::mapped_type;

  // Insert pairs
  auto pairs_begin = thrust::make_transform_iterator(
    thrust::counting_iterator<size_type>(0),
    cuda::proclaim_return_type<cuco::pair<Key, Value>>([] __device__(auto i) {
      return cuco::pair<Key, Value>{i, i};
    }));

  auto const initial_size = map.insert(pairs_begin, pairs_begin + num_keys);
  REQUIRE(initial_size == num_keys);  // all keys should be inserted

  // Query pairs have the same keys but different payloads
  auto query_pairs_begin = thrust::make_transform_iterator(
    thrust::counting_iterator<size_type>(0),
    cuda::proclaim_return_type<cuco::pair<Key, Value>>(
      [] __device__(auto i) { return cuco::pair<Key, Value>(i, i * 2); }));

  map.insert_or_assign(query_pairs_begin, query_pairs_begin + num_keys);

  auto const updated_size = map.size();
  // all keys are present in the map so the size shouldn't change
  REQUIRE(updated_size == initial_size);

  thrust::device_vector<Key> d_keys(num_keys);
  thrust::device_vector<Key> d_values(num_keys);
  map.retrieve_all(d_keys.begin(), d_values.begin());

  auto gold_values_begin = thrust::make_transform_iterator(
    thrust::counting_iterator<size_type>(0),
    cuda::proclaim_return_type<size_type>([] __device__(auto i) { return i * 2; }));

  thrust::sort(thrust::device, d_values.begin(), d_values.end());
  REQUIRE(cuco::test::equal(
    d_values.begin(), d_values.end(), gold_values_begin, thrust::equal_to<Value>{}));
}

TEMPLATE_TEST_CASE_SIG(
  "Insert or assign",
  "",
  ((typename Key, typename Value, cuco::test::probe_sequence Probe, int CGSize),
   Key,
   Value,
   Probe,
   CGSize),
  (int32_t, int32_t, cuco::test::probe_sequence::double_hashing, 1),
  (int32_t, int64_t, cuco::test::probe_sequence::double_hashing, 1),
  (int32_t, int32_t, cuco::test::probe_sequence::double_hashing, 2),
  (int32_t, int64_t, cuco::test::probe_sequence::double_hashing, 2),
  (int64_t, int32_t, cuco::test::probe_sequence::double_hashing, 1),
  (int64_t, int64_t, cuco::test::probe_sequence::double_hashing, 1),
  (int64_t, int32_t, cuco::test::probe_sequence::double_hashing, 2),
  (int64_t, int64_t, cuco::test::probe_sequence::double_hashing, 2),
  (int32_t, int32_t, cuco::test::probe_sequence::linear_probing, 1),
  (int32_t, int64_t, cuco::test::probe_sequence::linear_probing, 1),
  (int32_t, int32_t, cuco::test::probe_sequence::linear_probing, 2),
  (int32_t, int64_t, cuco::test::probe_sequence::linear_probing, 2),
  (int64_t, int32_t, cuco::test::probe_sequence::linear_probing, 1),
  (int64_t, int64_t, cuco::test::probe_sequence::linear_probing, 1),
  (int64_t, int32_t, cuco::test::probe_sequence::linear_probing, 2),
  (int64_t, int64_t, cuco::test::probe_sequence::linear_probing, 2))
{
  constexpr size_type num_keys{400};

  using probe = std::conditional_t<
    Probe == cuco::test::probe_sequence::linear_probing,
    cuco::linear_probing<CGSize, cuco::murmurhash3_32<Key>>,
    cuco::double_hashing<CGSize, cuco::murmurhash3_32<Key>, cuco::murmurhash3_32<Key>>>;

  auto map = cuco::static_map<Key,
                              Value,
                              cuco::extent<size_type>,
                              cuda::thread_scope_device,
                              thrust::equal_to<Key>,
                              probe,
                              cuco::cuda_allocator<std::byte>,
                              cuco::storage<2>>{
    num_keys, cuco::empty_key<Key>{-1}, cuco::empty_value<Value>{-1}};

  test_insert_or_assign(map, num_keys);
}
