#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, Jonas Hahnfeld, CERN.
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utils.hpp>

#include <cuco/static_map.cuh>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>

#include <catch2/catch_template_test_macros.hpp>

static constexpr int Iters = 10'000;

template <typename Ref>
__global__ void parallel_sum(Ref v)
{
  for (int i = 0; i < Iters; i++) {
#if __CUDA_ARCH__ < 700
    if constexpr (cuco::detail::is_packable<Ref::value_type>())
#endif
    {
      auto constexpr cg_size = Ref::cg_size;
      if constexpr (cg_size == 1) {
        auto [iter, inserted] = v.insert_and_find(cuco::pair{i, 1});
        // for debugging...
        // if (iter->second < 0) {
        //   asm("trap;");
        // }
        if (!inserted) {
          auto ref =
            cuda::atomic_ref<typename Ref::mapped_type, cuda::thread_scope_device>{iter->second};
          ref.fetch_add(1);
        }
      } else {
        auto const tile =
          cooperative_groups::tiled_partition<cg_size>(cooperative_groups::this_thread_block());
        auto [iter, inserted] = v.insert_and_find(tile, cuco::pair{i, 1});
        if (!inserted and tile.thread_rank() == 0) {
          auto ref =
            cuda::atomic_ref<typename Ref::mapped_type, cuda::thread_scope_device>{iter->second};
          ref.fetch_add(1);
        }
      }
    }
#if __CUDA_ARCH__ < 700
    else {
      v.insert(cuco::pair{i, gridDim.x * blockDim.x});
    }
#endif
  }
}

TEMPLATE_TEST_CASE_SIG(
  "static_map insert_and_find tests",
  "",
  ((typename Key, typename Value, cuco::test::probe_sequence Probe, int CGSize),
   Key,
   Value,
   Probe,
   CGSize),
  (int32_t, int32_t, cuco::test::probe_sequence::double_hashing, 1),
  (int32_t, int64_t, cuco::test::probe_sequence::double_hashing, 1),
  (int32_t, int32_t, cuco::test::probe_sequence::double_hashing, 2),
  (int32_t, int64_t, cuco::test::probe_sequence::double_hashing, 2),
  (int64_t, int32_t, cuco::test::probe_sequence::double_hashing, 1),
  (int64_t, int64_t, cuco::test::probe_sequence::double_hashing, 1),
  (int64_t, int32_t, cuco::test::probe_sequence::double_hashing, 2),
  (int64_t, int64_t, cuco::test::probe_sequence::double_hashing, 2),
  (int32_t, int32_t, cuco::test::probe_sequence::linear_probing, 1),
  (int32_t, int64_t, cuco::test::probe_sequence::linear_probing, 1),
  (int32_t, int32_t, cuco::test::probe_sequence::linear_probing, 2),
  (int32_t, int64_t, cuco::test::probe_sequence::linear_probing, 2),
  (int64_t, int32_t, cuco::test::probe_sequence::linear_probing, 1),
  (int64_t, int64_t, cuco::test::probe_sequence::linear_probing, 1),
  (int64_t, int32_t, cuco::test::probe_sequence::linear_probing, 2),
  (int64_t, int64_t, cuco::test::probe_sequence::linear_probing, 2))
{
  using probe =
    std::conditional_t<Probe == cuco::test::probe_sequence::linear_probing,
                       cuco::experimental::linear_probing<CGSize, cuco::murmurhash3_32<Key>>,
                       cuco::experimental::double_hashing<CGSize,
                                                          cuco::murmurhash3_32<Key>,
                                                          cuco::murmurhash3_32<Key>>>;

  auto map = cuco::experimental::static_map<Key,
                                            Value,
                                            cuco::experimental::extent<std::size_t>,
                                            cuda::thread_scope_device,
                                            thrust::equal_to<Key>,
                                            probe,
                                            cuco::cuda_allocator<std::byte>,
                                            cuco::experimental::storage<2>>{
    10 * Iters, cuco::empty_key<Key>{-1}, cuco::empty_value<Value>{-1}};

  static constexpr int Blocks  = 1024;
  static constexpr int Threads = 128;

  parallel_sum<<<Blocks, Threads>>>(
    map.ref(cuco::experimental::op::insert, cuco::experimental::op::insert_and_find));
  CUCO_CUDA_TRY(hipDeviceSynchronize());

  thrust::device_vector<Key> d_keys(Iters);
  thrust::device_vector<Value> d_values(Iters);

  thrust::sequence(thrust::device, d_keys.begin(), d_keys.end());
  map.find(d_keys.begin(), d_keys.end(), d_values.begin());

  REQUIRE(cuco::test::all_of(d_values.begin(), d_values.end(), [] __device__(Value v) {
    return v == (Blocks * Threads) / CGSize;
  }));
}
