#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, Jonas Hahnfeld, CERN.
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utils.hpp>

#include <cuco/static_map.cuh>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>

#include <catch2/catch.hpp>

static constexpr int Iters = 10'000;

template <typename View>
__global__ void parallel_sum(View v)
{
  for (int i = 0; i < Iters; i++) {
#if __CUDA_ARCH__ < 700
    if constexpr (cuco::detail::is_packable<View::value_type>())
#endif
    {
      auto [iter, inserted] = v.insert_and_find(thrust::make_pair(i, 1));
      // for debugging...
      // if (iter->second < 0) {
      //   asm("trap;");
      // }
      if (!inserted) { iter->second += 1; }
    }
#if __CUDA_ARCH__ < 700
    else {
      v.insert(thrust::make_pair(i, gridDim.x * blockDim.x));
    }
#endif
  }
}

TEMPLATE_TEST_CASE_SIG("Parallel insert-or-update",
                       "",
                       ((typename Key, typename Value), Key, Value),
                       (int32_t, int32_t),
                       (int32_t, int64_t),
                       (int64_t, int32_t),
                       (int64_t, int64_t))
{
  cuco::empty_key<Key> empty_key_sentinel{-1};
  cuco::empty_value<Value> empty_value_sentinel{-1};
  cuco::static_map<Key, Value> m(10 * Iters, empty_key_sentinel, empty_value_sentinel);

  static constexpr int Blocks  = 1024;
  static constexpr int Threads = 128;
  parallel_sum<<<Blocks, Threads>>>(m.get_device_mutable_view());
  hipDeviceSynchronize();

  thrust::device_vector<Key> d_keys(Iters);
  thrust::device_vector<Value> d_values(Iters);

  thrust::sequence(thrust::device, d_keys.begin(), d_keys.end());
  m.find(d_keys.begin(), d_keys.end(), d_values.begin());

  REQUIRE(cuco::test::all_of(
    d_values.begin(), d_values.end(), [] __device__(Value v) { return v == Blocks * Threads; }));
}
