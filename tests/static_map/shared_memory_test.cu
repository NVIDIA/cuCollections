#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <limits>

#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>

#include <catch2/catch.hpp>
#include <cuco/static_map.cuh>

#include <util.hpp>

enum class dist_type { UNIQUE, UNIFORM, GAUSSIAN };

template <dist_type Dist, typename Key, typename OutputIt>
static void generate_keys(OutputIt output_begin, OutputIt output_end)
{
  auto num_keys = std::distance(output_begin, output_end);

  std::random_device rd;
  std::mt19937 gen{rd()};

  switch (Dist) {
    case dist_type::UNIQUE:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = i;
      }
      break;
    case dist_type::UNIFORM:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(gen()));
      }
      break;
    case dist_type::GAUSSIAN:
      std::normal_distribution<> dg{1e9, 1e7};
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(dg(gen)));
      }
      break;
  }
}

template <typename MapType, int CAPACITY>
__global__ void shared_memory_test_kernel(
  typename MapType::device_view const* const device_views,
  typename MapType::device_view::key_type const* const insterted_keys,
  typename MapType::device_view::mapped_type const* const inserted_values,
  const size_t number_of_elements,
  bool* const keys_exist,
  bool* const keys_and_values_correct)
{
  // Each block processes one map
  const size_t map_id = blockIdx.x;
  const size_t offset = map_id * number_of_elements;

  __shared__ typename MapType::pair_atomic_type sm_buffer[CAPACITY];

  auto g = cg::this_thread_block();
  typename MapType::device_view sm_device_view =
    MapType::device_view::make_copy(g, sm_buffer, device_views[map_id]);

  for (int i = g.thread_rank(); i < number_of_elements; i += g.size()) {
    auto found_pair_it = sm_device_view.find(insterted_keys[offset + i]);

    if (found_pair_it != sm_device_view.end()) {
      keys_exist[offset + i] = true;
      if (found_pair_it->first == insterted_keys[offset + i] and
          found_pair_it->second == inserted_values[offset + i]) {
        keys_and_values_correct[offset + i] = true;
      } else {
        keys_and_values_correct[offset + i] = false;
      }
    } else {
      keys_exist[offset + i]              = false;
      keys_and_values_correct[offset + i] = true;
    }
  }
}

TEMPLATE_TEST_CASE_SIG("Shared memory static map",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int64_t, dist_type::UNIQUE),
                       (int32_t, dist_type::UNIFORM),
                       (int64_t, dist_type::UNIFORM),
                       (int32_t, dist_type::GAUSSIAN),
                       (int64_t, dist_type::GAUSSIAN))
{
  using KeyType                = T;
  using ValueType              = T;
  using MapType                = cuco::static_map<KeyType, ValueType>;
  using DeviceViewType         = typename MapType::device_view;
  using DeviceViewIteratorType = typename DeviceViewType::iterator;

  constexpr std::size_t number_of_maps  = 1000;
  constexpr std::size_t elements_in_map = 500;
  constexpr std::size_t map_capacity    = 2 * elements_in_map;

  // one array for all maps, first elements_in_map element belong to map 0, second to map 1 and so
  // on
  std::vector<KeyType> h_keys(number_of_maps * elements_in_map);
  std::vector<ValueType> h_values(number_of_maps * elements_in_map);
  std::vector<cuco::pair_type<KeyType, ValueType>> h_pairs(number_of_maps * elements_in_map);

  // using std::unique_ptr because static_map does not have copy/move constructor/assignment
  // operator yet
  std::vector<std::unique_ptr<MapType>> maps;

  for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
    const std::size_t offset = map_id * elements_in_map;

    generate_keys<Dist, KeyType>(h_keys.begin() + offset,
                                 h_keys.begin() + offset + elements_in_map);

    for (std::size_t i = 0; i < elements_in_map; ++i) {
      KeyType key                = h_keys[offset + i];
      ValueType val              = key < std::numeric_limits<KeyType>::max() ? key + 1 : 0;
      h_values[offset + i]       = val;
      h_pairs[offset + i].first  = key;
      h_pairs[offset + i].second = val;
    }

    maps.push_back(std::make_unique<MapType>(map_capacity, -1, -1));
  }

  thrust::device_vector<KeyType> d_keys(h_keys);
  thrust::device_vector<ValueType> d_values(h_values);
  thrust::device_vector<cuco::pair_type<KeyType, ValueType>> d_pairs(h_pairs);

  SECTION("Keys are all found after insertion.")
  {
    std::vector<DeviceViewType> h_device_views;
    for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
      const std::size_t offset = map_id * elements_in_map;

      MapType* map = maps[map_id].get();
      map->insert(d_pairs.begin() + offset, d_pairs.begin() + offset + elements_in_map);
      h_device_views.push_back(map->get_device_view());
    }
    thrust::device_vector<DeviceViewType> d_device_views(h_device_views);

    thrust::device_vector<bool> d_keys_exist(number_of_maps * elements_in_map);
    thrust::device_vector<bool> d_keys_and_values_correct(number_of_maps * elements_in_map);

    shared_memory_test_kernel<MapType, map_capacity>
      <<<number_of_maps, 64>>>(d_device_views.data().get(),
                               d_keys.data().get(),
                               d_values.data().get(),
                               elements_in_map,
                               d_keys_exist.data().get(),
                               d_keys_and_values_correct.data().get());

    REQUIRE(d_keys_exist.size() == d_keys_and_values_correct.size());
    auto zip = thrust::make_zip_iterator(
      thrust::make_tuple(d_keys_exist.begin(), d_keys_and_values_correct.begin()));

    REQUIRE(all_of(zip, zip + d_keys_exist.size(), [] __device__(auto const& z) {
      return thrust::get<0>(z) and thrust::get<1>(z);
    }));
  }

  SECTION("No key is found before insertion.")
  {
    std::vector<DeviceViewType> h_device_views;
    for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
      h_device_views.push_back(maps[map_id].get()->get_device_view());
    }
    thrust::device_vector<DeviceViewType> d_device_views(h_device_views);

    thrust::device_vector<bool> d_keys_exist(number_of_maps * elements_in_map);
    thrust::device_vector<bool> d_keys_and_values_correct(number_of_maps * elements_in_map);

    shared_memory_test_kernel<MapType, map_capacity>
      <<<number_of_maps, 64>>>(d_device_views.data().get(),
                               d_keys.data().get(),
                               d_values.data().get(),
                               elements_in_map,
                               d_keys_exist.data().get(),
                               d_keys_and_values_correct.data().get());

    REQUIRE(none_of(d_keys_exist.begin(), d_keys_exist.end(), [] __device__(const bool key_found) {
      return key_found;
    }));
  }
}

template <typename K, typename V, std::size_t N>
__global__ void shared_memory_hash_table_kernel(bool* key_found)
{
  namespace cg   = cooperative_groups;
  using map_type = typename cuco::static_map<K, V, cuda::thread_scope_block>::device_mutable_view;
  using find_map_type = typename cuco::static_map<K, V, cuda::thread_scope_block>::device_view;
  __shared__ typename map_type::slot_type slots[N];
  auto map = map_type::make_from_uninitialized_slots(cg::this_thread_block(), &slots[0], N, -1, -1);

  auto g            = cg::this_thread_block();
  std::size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  int rank          = g.thread_rank();

  // insert {thread_rank, thread_rank} for each thread in thread-block
  map.insert(cuco::pair<int, int>(rank, rank));
  g.sync();

  auto find_map       = find_map_type(map);
  auto retrieved_pair = find_map.find(rank);
  if (retrieved_pair != find_map.end() && retrieved_pair->second == rank) {
    key_found[index] = true;
  }
}

TEMPLATE_TEST_CASE("Shared memory slots.", "", int32_t)
{
  constexpr std::size_t N = 256;
  thrust::device_vector<bool> key_found(N, false);
  shared_memory_hash_table_kernel<TestType, TestType, N><<<8, 32>>>(key_found.data().get());

  REQUIRE(all_of(key_found.begin(), key_found.end(), thrust::identity<bool>{}));
}
