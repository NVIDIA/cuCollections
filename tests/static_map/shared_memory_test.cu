#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utils.hpp>

#include <cuco/static_map.cuh>

#include <thrust/device_vector.h>

#include <catch2/catch.hpp>

#include <limits>

template <typename MapType, int CAPACITY>
__global__ void shared_memory_test_kernel(
  typename MapType::device_view const* const device_views,
  typename MapType::device_view::key_type const* const insterted_keys,
  typename MapType::device_view::mapped_type const* const inserted_values,
  const size_t number_of_elements,
  bool* const keys_exist,
  bool* const keys_and_values_correct)
{
  // Each block processes one map
  const size_t map_id = blockIdx.x;
  const size_t offset = map_id * number_of_elements;

  __shared__ typename MapType::pair_atomic_type sm_buffer[CAPACITY];

  auto g = cuco::test::cg::this_thread_block();
  typename MapType::device_view sm_device_view =
    MapType::device_view::make_copy(g, sm_buffer, device_views[map_id]);

  for (int i = g.thread_rank(); i < number_of_elements; i += g.size()) {
    auto found_pair_it = sm_device_view.find(insterted_keys[offset + i]);

    if (found_pair_it != sm_device_view.end()) {
      keys_exist[offset + i] = true;
      if (found_pair_it->first == insterted_keys[offset + i] and
          found_pair_it->second == inserted_values[offset + i]) {
        keys_and_values_correct[offset + i] = true;
      } else {
        keys_and_values_correct[offset + i] = false;
      }
    } else {
      keys_exist[offset + i]              = false;
      keys_and_values_correct[offset + i] = true;
    }
  }
}

TEMPLATE_TEST_CASE_SIG("Shared memory static map",
                       "",
                       ((typename Key, typename Value), Key, Value),
                       (int32_t, int32_t),
                       (int32_t, int64_t),
                       (int64_t, int32_t),
                       (int64_t, int64_t))
{
  using MapType                = cuco::static_map<Key, Value>;
  using DeviceViewType         = typename MapType::device_view;
  using DeviceViewIteratorType = typename DeviceViewType::iterator;

  constexpr std::size_t number_of_maps  = 1000;
  constexpr std::size_t elements_in_map = 500;
  constexpr std::size_t map_capacity    = 2 * elements_in_map;

  // one array for all maps, first elements_in_map element belong to map 0, second to map 1 and so
  // on
  thrust::device_vector<Key> d_keys(number_of_maps * elements_in_map);
  thrust::device_vector<Value> d_values(number_of_maps * elements_in_map);

  thrust::sequence(thrust::device, d_keys.begin(), d_keys.end());
  thrust::sequence(thrust::device, d_values.begin(), d_values.end(), 1);

  // using std::unique_ptr because static_map does not have copy/move constructor/assignment
  // operator yet
  std::vector<std::unique_ptr<MapType>> maps;
  for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
    maps.push_back(std::make_unique<MapType>(
      map_capacity, cuco::sentinel::empty_key<Key>{-1}, cuco::sentinel::empty_value<Value>{-1}));
  }

  thrust::device_vector<bool> d_keys_exist(number_of_maps * elements_in_map);
  thrust::device_vector<bool> d_keys_and_values_correct(number_of_maps * elements_in_map);

  SECTION("Keys are all found after insertion.")
  {
    auto pairs_begin =
      thrust::make_zip_iterator(thrust::make_tuple(d_keys.begin(), d_values.begin()));
    std::vector<DeviceViewType> h_device_views;
    for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
      const std::size_t offset = map_id * elements_in_map;

      MapType* map = maps[map_id].get();
      map->insert(pairs_begin + offset, pairs_begin + offset + elements_in_map);
      h_device_views.push_back(map->get_device_view());
    }
    thrust::device_vector<DeviceViewType> d_device_views(h_device_views);

    shared_memory_test_kernel<MapType, map_capacity>
      <<<number_of_maps, 64>>>(d_device_views.data().get(),
                               d_keys.data().get(),
                               d_values.data().get(),
                               elements_in_map,
                               d_keys_exist.data().get(),
                               d_keys_and_values_correct.data().get());

    REQUIRE(d_keys_exist.size() == d_keys_and_values_correct.size());
    auto zip = thrust::make_zip_iterator(
      thrust::make_tuple(d_keys_exist.begin(), d_keys_and_values_correct.begin()));

    REQUIRE(cuco::test::all_of(zip, zip + d_keys_exist.size(), [] __device__(auto const& z) {
      return thrust::get<0>(z) and thrust::get<1>(z);
    }));
  }

  SECTION("No key is found before insertion.")
  {
    std::vector<DeviceViewType> h_device_views;
    for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
      h_device_views.push_back(maps[map_id].get()->get_device_view());
    }
    thrust::device_vector<DeviceViewType> d_device_views(h_device_views);

    shared_memory_test_kernel<MapType, map_capacity>
      <<<number_of_maps, 64>>>(d_device_views.data().get(),
                               d_keys.data().get(),
                               d_values.data().get(),
                               elements_in_map,
                               d_keys_exist.data().get(),
                               d_keys_and_values_correct.data().get());

    REQUIRE(cuco::test::none_of(d_keys_exist.begin(),
                                d_keys_exist.end(),
                                [] __device__(const bool key_found) { return key_found; }));
  }
}

template <typename K, typename V, std::size_t N>
__global__ void shared_memory_hash_table_kernel(bool* key_found)
{
  namespace cg   = cooperative_groups;
  using map_type = typename cuco::static_map<K, V, cuda::thread_scope_block>::device_mutable_view;
  using find_map_type = typename cuco::static_map<K, V, cuda::thread_scope_block>::device_view;
  __shared__ typename map_type::slot_type slots[N];
  auto map = map_type::make_from_uninitialized_slots(cg::this_thread_block(),
                                                     &slots[0],
                                                     N,
                                                     cuco::sentinel::empty_key<K>{-1},
                                                     cuco::sentinel::empty_value<V>{-1});

  auto g            = cg::this_thread_block();
  std::size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  int rank          = g.thread_rank();

  // insert {thread_rank, thread_rank} for each thread in thread-block
  map.insert(cuco::pair<int, int>(rank, rank));
  g.sync();

  auto find_map       = find_map_type(map);
  auto retrieved_pair = find_map.find(rank);
  if (retrieved_pair != find_map.end() && retrieved_pair->second == rank) {
    key_found[index] = true;
  }
}

TEMPLATE_TEST_CASE("Shared memory slots.", "", int32_t)
{
  constexpr std::size_t N = 256;
  thrust::device_vector<bool> key_found(N, false);
  shared_memory_hash_table_kernel<TestType, TestType, N><<<8, 32>>>(key_found.data().get());

  REQUIRE(cuco::test::all_of(key_found.begin(), key_found.end(), thrust::identity<bool>{}));
}
