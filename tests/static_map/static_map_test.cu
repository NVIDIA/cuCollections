#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <limits>

#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>

#include <catch2/catch.hpp>

#include <cuco/static_map.cuh>

namespace {
namespace cg = cooperative_groups;

// User-defined logical algorithms to reduce compilation time
template <typename Iterator, typename Predicate>
bool all_of(Iterator begin, Iterator end, Predicate p, hipStream_t stream = nullptr)
{
  auto size = thrust::distance(begin, end);
  auto out  = thrust::count_if(thrust::cuda::par.on(stream), begin, end, p);
  hipStreamSynchronize(stream);
  return size == out;
}

template <typename Iterator, typename Predicate>
bool any_of(Iterator begin, Iterator end, Predicate p)
{
  return thrust::count_if(begin, end, p) > 0;
}

template <typename Iterator, typename Predicate>
bool none_of(Iterator begin, Iterator end, Predicate p)
{
  return not all_of(begin, end, p);
}
}  // namespace

enum class dist_type { UNIQUE, UNIFORM, GAUSSIAN };

template <dist_type Dist, typename Key, typename OutputIt>
static void generate_keys(OutputIt output_begin, OutputIt output_end)
{
  auto num_keys = std::distance(output_begin, output_end);

  std::random_device rd;
  std::mt19937 gen{rd()};

  switch (Dist) {
    case dist_type::UNIQUE:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = i;
      }
      break;
    case dist_type::UNIFORM:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(gen()));
      }
      break;
    case dist_type::GAUSSIAN:
      std::normal_distribution<> dg{1e9, 1e7};
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(dg(gen)));
      }
      break;
  }
}

// User-defined key type
template <typename T>
struct key_pair_type {
  T a;
  T b;

  __host__ __device__ key_pair_type() {}
  __host__ __device__ key_pair_type(T x) : a{x}, b{x} {}

  // Device equality operator is mandatory due to libcudacxx bug:
  // https://github.com/NVIDIA/libcudacxx/issues/223
  __device__ bool operator==(key_pair_type const& other) const
  {
    return a == other.a and b == other.b;
  }
};

// User-defined key type
template <typename T>
struct large_key_type {
  T a;
  T b;
  T c;

  __host__ __device__ large_key_type() {}
  __host__ __device__ large_key_type(T x) : a{x}, b{x}, c{x} {}

  // Device equality operator is mandatory due to libcudacxx bug:
  // https://github.com/NVIDIA/libcudacxx/issues/223
  __device__ bool operator==(large_key_type const& other) const
  {
    return a == other.a and b == other.b and c == other.c;
  }
};

// User-defined value type
template <typename T>
struct value_pair_type {
  T f;
  T s;

  __host__ __device__ value_pair_type() {}
  __host__ __device__ value_pair_type(T x) : f{x}, s{x} {}

  __device__ bool operator==(value_pair_type const& other) const
  {
    return f == other.f and s == other.s;
  }
};

// User-defined device hasher
struct hash_custom_key {
  template <typename custom_type>
  __device__ uint32_t operator()(custom_type k)
  {
    return k.a;
  };
};

// User-defined device key equality
struct custom_key_equals {
  template <typename custom_type>
  __device__ bool operator()(custom_type lhs, custom_type rhs)
  {
    return std::tie(lhs.a, lhs.b) == std::tie(rhs.a, rhs.b);
  }
};

#define SIZE 10
__device__ int A[SIZE];

template <typename T>
struct custom_equals {
  __device__ bool operator()(T lhs, T rhs) { return A[lhs] == A[rhs]; }
};

TEMPLATE_TEST_CASE_SIG("User defined key and value type",
                       "",
                       ((typename Key, typename Value), Key, Value),
#ifndef CUCO_NO_INDEPENDENT_THREADS  // Key type larger than 8B only supported for sm_70 and up
                       (key_pair_type<int64_t>, value_pair_type<int32_t>),
                       (key_pair_type<int64_t>, value_pair_type<int64_t>),
                       (large_key_type<int32_t>, value_pair_type<int32_t>),
#endif
                       (key_pair_type<int32_t>, value_pair_type<int32_t>))
{
  auto const sentinel_key   = Key{-1};
  auto const sentinel_value = Value{-1};

  constexpr std::size_t num      = 100;
  constexpr std::size_t capacity = num * 2;
  cuco::static_map<Key, Value> map{capacity, sentinel_key, sentinel_value};

  thrust::device_vector<Key> insert_keys(num);
  thrust::device_vector<Value> insert_values(num);

  thrust::transform(thrust::device,
                    thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(num),
                    insert_keys.begin(),
                    [] __device__(auto i) { return Key{i}; });

  thrust::transform(thrust::device,
                    thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(num),
                    insert_values.begin(),
                    [] __device__(auto i) { return Value{i}; });

  auto insert_pairs =
    thrust::make_zip_iterator(thrust::make_tuple(insert_keys.begin(), insert_values.begin()));

  SECTION("All inserted keys-value pairs should be correctly recovered during find")
  {
    thrust::device_vector<Value> found_values(num);
    map.insert(insert_pairs, insert_pairs + num, hash_custom_key{}, custom_key_equals{});

    REQUIRE(num == map.get_size());

    map.find(insert_keys.begin(),
             insert_keys.end(),
             found_values.begin(),
             hash_custom_key{},
             custom_key_equals{});

    REQUIRE(thrust::equal(thrust::device,
                          insert_values.begin(),
                          insert_values.end(),
                          found_values.begin(),
                          [] __device__(Value lhs, Value rhs) {
                            return std::tie(lhs.f, lhs.s) == std::tie(rhs.f, rhs.s);
                          }));
  }

  SECTION("All inserted keys-value pairs should be contained")
  {
    thrust::device_vector<bool> contained(num);
    map.insert(insert_pairs, insert_pairs + num, hash_custom_key{}, custom_key_equals{});
    map.contains(insert_keys.begin(),
                 insert_keys.end(),
                 contained.begin(),
                 hash_custom_key{},
                 custom_key_equals{});
    REQUIRE(all_of(contained.begin(), contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("All conditionally inserted keys-value pairs should be contained")
  {
    thrust::device_vector<bool> contained(num);
    map.insert_if(
      insert_pairs,
      insert_pairs + num,
      thrust::counting_iterator<int>(0),
      [] __device__(auto const& key) { return (key % 2) == 0; },
      hash_custom_key{},
      custom_key_equals{});

    REQUIRE(num / 2 == map.get_size());

    map.contains(insert_keys.begin(),
                 insert_keys.end(),
                 contained.begin(),
                 hash_custom_key{},
                 custom_key_equals{});

    REQUIRE(thrust::equal(thrust::device,
                          contained.begin(),
                          contained.end(),
                          thrust::counting_iterator<int>(0),
                          [] __device__(auto const& idx_contained, auto const& idx) {
                            return ((idx % 2) == 0) == idx_contained;
                          }));
  }

  SECTION("Non-inserted keys-value pairs should not be contained")
  {
    thrust::device_vector<bool> contained(num);
    map.contains(insert_keys.begin(),
                 insert_keys.end(),
                 contained.begin(),
                 hash_custom_key{},
                 custom_key_equals{});
    REQUIRE(
      none_of(contained.begin(), contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("All inserted keys-value pairs should be contained")
  {
    thrust::device_vector<bool> contained(num);
    map.insert(insert_pairs, insert_pairs + num, hash_custom_key{}, custom_key_equals{});
    auto view = map.get_device_view();
    REQUIRE(all_of(
      insert_pairs, insert_pairs + num, [view] __device__(cuco::pair_type<Key, Value> const& pair) {
        return view.contains(pair.first, hash_custom_key{}, custom_key_equals{});
      }));
  }

  SECTION("Inserting unique keys should return insert success.")
  {
    auto m_view = map.get_device_mutable_view();
    REQUIRE(all_of(insert_pairs,
                   insert_pairs + num,
                   [m_view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                     return m_view.insert(pair, hash_custom_key{}, custom_key_equals{});
                   }));
  }

  SECTION("Cannot find any key in an empty hash map")
  {
    SECTION("non-const view")
    {
      auto view = map.get_device_view();
      REQUIRE(all_of(insert_pairs,
                     insert_pairs + num,
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       return view.find(pair.first, hash_custom_key{}, custom_key_equals{}) ==
                              view.end();
                     }));
    }

    SECTION("const view")
    {
      auto const view = map.get_device_view();
      REQUIRE(all_of(insert_pairs,
                     insert_pairs + num,
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) {
                       return view.find(pair.first, hash_custom_key{}, custom_key_equals{}) ==
                              view.end();
                     }));
    }
  }
}

TEMPLATE_TEST_CASE_SIG("Key comparison against sentinel",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int64_t, dist_type::UNIQUE))
{
  using Key   = T;
  using Value = T;

  constexpr std::size_t num_keys{SIZE};
  cuco::static_map<Key, Value> map{SIZE * 2, -1, -1};

  auto m_view = map.get_device_mutable_view();
  auto view   = map.get_device_view();

  std::vector<Key> h_keys(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_keys<Dist, Key>(h_keys.begin(), h_keys.end());

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
  }

  int h_A[SIZE];
  for (int i = 0; i < SIZE; i++) {
    h_A[i] = i;
  }
  hipMemcpyToSymbol(HIP_SYMBOL(A), h_A, SIZE * sizeof(int));

  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);

  SECTION(
    "Tests of non-CG insert: The custom `key_equal` can never be used to compare against sentinel")
  {
    REQUIRE(all_of(d_pairs.begin(),
                   d_pairs.end(),
                   [m_view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                     return m_view.insert(
                       pair, cuco::detail::MurmurHash3_32<Key>{}, custom_equals<Key>{});
                   }));
  }

  SECTION(
    "Tests of CG insert: The custom `key_equal` can never be used to compare against sentinel")
  {
    map.insert(
      d_pairs.begin(), d_pairs.end(), cuco::detail::MurmurHash3_32<Key>{}, custom_equals<Key>{});
    // All keys inserted via custom `key_equal` should be found
    REQUIRE(all_of(
      d_pairs.begin(), d_pairs.end(), [view] __device__(cuco::pair_type<Key, Value> const& pair) {
        auto const found = view.find(pair.first);
        return (found != view.end()) and
               (found->first.load() == pair.first and found->second.load() == pair.second);
      }));
  }
}

TEMPLATE_TEST_CASE_SIG("Unique sequence of keys",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int64_t, dist_type::UNIQUE),
                       (int32_t, dist_type::UNIFORM),
                       (int64_t, dist_type::UNIFORM),
                       (int32_t, dist_type::GAUSSIAN),
                       (int64_t, dist_type::GAUSSIAN))
{
  using Key   = T;
  using Value = T;

  constexpr std::size_t num_keys{500'000};
  cuco::static_map<Key, Value> map{1'000'000, -1, -1};

  auto m_view = map.get_device_mutable_view();
  auto view   = map.get_device_view();

  std::vector<Key> h_keys(num_keys);
  std::vector<Value> h_values(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_keys<Dist, Key>(h_keys.begin(), h_keys.end());

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
    h_values[i]       = val;
  }

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<Value> d_values(h_values);
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);
  thrust::device_vector<Value> d_results(num_keys);
  thrust::device_vector<bool> d_contained(num_keys);

  // bulk function test cases
  SECTION("All inserted keys-value pairs should be correctly recovered during find")
  {
    map.insert(d_pairs.begin(), d_pairs.end());
    map.find(d_keys.begin(), d_keys.end(), d_results.begin());
    auto zip = thrust::make_zip_iterator(thrust::make_tuple(d_results.begin(), d_values.begin()));

    REQUIRE(all_of(zip, zip + num_keys, [] __device__(auto const& p) {
      return thrust::get<0>(p) == thrust::get<1>(p);
    }));
  }

  SECTION("All inserted keys-value pairs should be contained")
  {
    map.insert(d_pairs.begin(), d_pairs.end());
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(
      all_of(d_contained.begin(), d_contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("Non-inserted keys-value pairs should not be contained")
  {
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(
      none_of(d_contained.begin(), d_contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("Inserting unique keys should return insert success.")
  {
    if (Dist == dist_type::UNIQUE) {
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [m_view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       return m_view.insert(pair);
                     }));
    }
  }

  SECTION("Cannot find any key in an empty hash map with non-const view")
  {
    SECTION("non-const view")
    {
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       return view.find(pair.first) == view.end();
                     }));
    }
    SECTION("const view")
    {
      REQUIRE(all_of(
        d_pairs.begin(), d_pairs.end(), [view] __device__(cuco::pair_type<Key, Value> const& pair) {
          return view.find(pair.first) == view.end();
        }));
    }
  }

  SECTION("Keys are all found after inserting many keys.")
  {
    // Bulk insert keys
    thrust::for_each(thrust::device,
                     d_pairs.begin(),
                     d_pairs.end(),
                     [m_view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       m_view.insert(pair);
                     });

    SECTION("non-const view")
    {
      // All keys should be found
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       auto const found = view.find(pair.first);
                       return (found != view.end()) and (found->first.load() == pair.first and
                                                         found->second.load() == pair.second);
                     }));
    }
    SECTION("const view")
    {
      // All keys should be found
      REQUIRE(all_of(
        d_pairs.begin(), d_pairs.end(), [view] __device__(cuco::pair_type<Key, Value> const& pair) {
          auto const found = view.find(pair.first);
          return (found != view.end()) and
                 (found->first.load() == pair.first and found->second.load() == pair.second);
        }));
    }
  }
}

TEMPLATE_TEST_CASE_SIG("Unique sequence of keys on given stream",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int64_t, dist_type::UNIFORM),
                       (int32_t, dist_type::GAUSSIAN))
{
  using Key   = T;
  using Value = T;

  hipStream_t stream;
  hipStreamCreate(&stream);

  constexpr std::size_t num_keys{500'000};
  cuco::static_map<Key, Value> map{1'000'000, -1, -1, cuco::cuda_allocator<char>{}, stream};

  auto m_view = map.get_device_mutable_view();
  auto view   = map.get_device_view();

  std::vector<Key> h_keys(num_keys);
  std::vector<Value> h_values(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_keys<Dist, Key>(h_keys.begin(), h_keys.end());

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
    h_values[i]       = val;
  }

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<Value> d_values(h_values);
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);
  thrust::device_vector<Value> d_results(num_keys);
  thrust::device_vector<bool> d_contained(num_keys);

  auto hash_fn  = cuco::detail::MurmurHash3_32<Key>{};
  auto equal_fn = thrust::equal_to<Value>{};

  // bulk function test cases
  SECTION("All inserted keys-value pairs should be correctly recovered during find")
  {
    map.insert(d_pairs.begin(), d_pairs.end(), hash_fn, equal_fn, stream);
    map.find(d_keys.begin(), d_keys.end(), d_results.begin(), hash_fn, equal_fn, stream);
    // hipStreamSynchronize(stream);
    auto zip = thrust::make_zip_iterator(thrust::make_tuple(d_results.begin(), d_values.begin()));

    REQUIRE(all_of(
      zip,
      zip + num_keys,
      [] __device__(auto const& p) { return thrust::get<0>(p) == thrust::get<1>(p); },
      stream));
  }
  SECTION("All inserted keys-value pairs should be contained")
  {
    map.insert(d_pairs.begin(), d_pairs.end(), hash_fn, equal_fn, stream);
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin(), hash_fn, equal_fn, stream);

    REQUIRE(all_of(
      d_contained.begin(), d_contained.end(), [] __device__(bool const& b) { return b; }, stream));
  }

  hipStreamDestroy(stream);
}

template <typename MapType, int CAPACITY>
__global__ void shared_memory_test_kernel(
  typename MapType::device_view const* const device_views,
  typename MapType::device_view::key_type const* const insterted_keys,
  typename MapType::device_view::mapped_type const* const inserted_values,
  const size_t number_of_elements,
  bool* const keys_exist,
  bool* const keys_and_values_correct)
{
  // Each block processes one map
  const size_t map_id = blockIdx.x;
  const size_t offset = map_id * number_of_elements;

  __shared__ typename MapType::pair_atomic_type sm_buffer[CAPACITY];

  auto g = cg::this_thread_block();
  typename MapType::device_view sm_device_view =
    MapType::device_view::make_copy(g, sm_buffer, device_views[map_id]);

  for (int i = g.thread_rank(); i < number_of_elements; i += g.size()) {
    auto found_pair_it = sm_device_view.find(insterted_keys[offset + i]);

    if (found_pair_it != sm_device_view.end()) {
      keys_exist[offset + i] = true;
      if (found_pair_it->first == insterted_keys[offset + i] and
          found_pair_it->second == inserted_values[offset + i]) {
        keys_and_values_correct[offset + i] = true;
      } else {
        keys_and_values_correct[offset + i] = false;
      }
    } else {
      keys_exist[offset + i]              = false;
      keys_and_values_correct[offset + i] = true;
    }
  }
}

TEMPLATE_TEST_CASE_SIG("Shared memory static map",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int64_t, dist_type::UNIQUE),
                       (int32_t, dist_type::UNIFORM),
                       (int64_t, dist_type::UNIFORM),
                       (int32_t, dist_type::GAUSSIAN),
                       (int64_t, dist_type::GAUSSIAN))
{
  using KeyType                = T;
  using ValueType              = T;
  using MapType                = cuco::static_map<KeyType, ValueType>;
  using DeviceViewType         = typename MapType::device_view;
  using DeviceViewIteratorType = typename DeviceViewType::iterator;

  constexpr std::size_t number_of_maps  = 1000;
  constexpr std::size_t elements_in_map = 500;
  constexpr std::size_t map_capacity    = 2 * elements_in_map;

  // one array for all maps, first elements_in_map element belong to map 0, second to map 1 and so
  // on
  std::vector<KeyType> h_keys(number_of_maps * elements_in_map);
  std::vector<ValueType> h_values(number_of_maps * elements_in_map);
  std::vector<cuco::pair_type<KeyType, ValueType>> h_pairs(number_of_maps * elements_in_map);

  // using std::unique_ptr because static_map does not have copy/move constructor/assignment
  // operator yet
  std::vector<std::unique_ptr<MapType>> maps;

  for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
    const std::size_t offset = map_id * elements_in_map;

    generate_keys<Dist, KeyType>(h_keys.begin() + offset,
                                 h_keys.begin() + offset + elements_in_map);

    for (std::size_t i = 0; i < elements_in_map; ++i) {
      KeyType key                = h_keys[offset + i];
      ValueType val              = key < std::numeric_limits<KeyType>::max() ? key + 1 : 0;
      h_values[offset + i]       = val;
      h_pairs[offset + i].first  = key;
      h_pairs[offset + i].second = val;
    }

    maps.push_back(std::make_unique<MapType>(map_capacity, -1, -1));
  }

  thrust::device_vector<KeyType> d_keys(h_keys);
  thrust::device_vector<ValueType> d_values(h_values);
  thrust::device_vector<cuco::pair_type<KeyType, ValueType>> d_pairs(h_pairs);

  SECTION("Keys are all found after insertion.")
  {
    std::vector<DeviceViewType> h_device_views;
    for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
      const std::size_t offset = map_id * elements_in_map;

      MapType* map = maps[map_id].get();
      map->insert(d_pairs.begin() + offset, d_pairs.begin() + offset + elements_in_map);
      h_device_views.push_back(map->get_device_view());
    }
    thrust::device_vector<DeviceViewType> d_device_views(h_device_views);

    thrust::device_vector<bool> d_keys_exist(number_of_maps * elements_in_map);
    thrust::device_vector<bool> d_keys_and_values_correct(number_of_maps * elements_in_map);

    shared_memory_test_kernel<MapType, map_capacity>
      <<<number_of_maps, 64>>>(d_device_views.data().get(),
                               d_keys.data().get(),
                               d_values.data().get(),
                               elements_in_map,
                               d_keys_exist.data().get(),
                               d_keys_and_values_correct.data().get());

    REQUIRE(d_keys_exist.size() == d_keys_and_values_correct.size());
    auto zip = thrust::make_zip_iterator(
      thrust::make_tuple(d_keys_exist.begin(), d_keys_and_values_correct.begin()));

    REQUIRE(all_of(zip, zip + d_keys_exist.size(), [] __device__(auto const& z) {
      return thrust::get<0>(z) and thrust::get<1>(z);
    }));
  }

  SECTION("No key is found before insertion.")
  {
    std::vector<DeviceViewType> h_device_views;
    for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
      h_device_views.push_back(maps[map_id].get()->get_device_view());
    }
    thrust::device_vector<DeviceViewType> d_device_views(h_device_views);

    thrust::device_vector<bool> d_keys_exist(number_of_maps * elements_in_map);
    thrust::device_vector<bool> d_keys_and_values_correct(number_of_maps * elements_in_map);

    shared_memory_test_kernel<MapType, map_capacity>
      <<<number_of_maps, 64>>>(d_device_views.data().get(),
                               d_keys.data().get(),
                               d_values.data().get(),
                               elements_in_map,
                               d_keys_exist.data().get(),
                               d_keys_and_values_correct.data().get());

    REQUIRE(none_of(d_keys_exist.begin(), d_keys_exist.end(), [] __device__(const bool key_found) {
      return key_found;
    }));
  }
}

template <typename K, typename V, std::size_t N>
__global__ void shared_memory_hash_table_kernel(bool* key_found)
{
  namespace cg   = cooperative_groups;
  using map_type = typename cuco::static_map<K, V, cuda::thread_scope_block>::device_mutable_view;
  using find_map_type = typename cuco::static_map<K, V, cuda::thread_scope_block>::device_view;
  __shared__ typename map_type::slot_type slots[N];
  auto map = map_type::make_from_uninitialized_slots(cg::this_thread_block(), &slots[0], N, -1, -1);

  auto g            = cg::this_thread_block();
  std::size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  int rank          = g.thread_rank();

  // insert {thread_rank, thread_rank} for each thread in thread-block
  map.insert(cuco::pair<int, int>(rank, rank));
  g.sync();

  auto find_map       = find_map_type(map);
  auto retrieved_pair = find_map.find(rank);
  if (retrieved_pair != find_map.end() && retrieved_pair->second == rank) {
    key_found[index] = true;
  }
}

TEMPLATE_TEST_CASE("Shared memory slots.", "", int32_t)
{
  constexpr std::size_t N = 256;
  thrust::device_vector<bool> key_found(N, false);
  shared_memory_hash_table_kernel<TestType, TestType, N><<<8, 32>>>(key_found.data().get());

  REQUIRE(all_of(key_found.begin(), key_found.end(), thrust::identity<bool>{}));
}
