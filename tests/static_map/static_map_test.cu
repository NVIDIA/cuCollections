#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>
#include <algorithm>
#include <catch2/catch.hpp>
#include <cuco/static_map.cuh>
#include <limits>
#include <util.hpp>

enum class dist_type { UNIQUE, UNIFORM, GAUSSIAN };

template <dist_type Dist, typename Key, typename OutputIt>
static void generate_keys(OutputIt output_begin, OutputIt output_end)
{
  auto num_keys = std::distance(output_begin, output_end);

  std::random_device rd;
  std::mt19937 gen{rd()};

  switch (Dist) {
    case dist_type::UNIQUE:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = i;
      }
      break;
    case dist_type::UNIFORM:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(gen()));
      }
      break;
    case dist_type::GAUSSIAN:
      std::normal_distribution<> dg{1e9, 1e7};
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(dg(gen)));
      }
      break;
  }
}

// User-defined key type
// Need to specify alignment to WAR libcu++ bug where cuda::atomic fails for underaligned types:
// https://github.com/NVIDIA/libcudacxx/issues/160
struct alignas(8) key_pair {
  int32_t a;
  int32_t b;
};

struct hash_key_pair {
  __device__ uint32_t operator()(key_pair k) { return k.a; };
};

struct key_pair_equals {
  __device__ bool operator()(key_pair lhs, key_pair rhs)
  {
    return std::tie(lhs.a, lhs.b) == std::tie(rhs.a, rhs.b);
  }
};

struct alignas(8) value_pair {
  int32_t f;
  int32_t s;
};

TEST_CASE("User defined key and value type", "")
{
  using Key   = key_pair;
  using Value = value_pair;

  auto constexpr sentinel_key   = Key{-1, -1};
  auto constexpr sentinel_value = Value{-1, -1};

  constexpr std::size_t num_pairs = 100;
  constexpr std::size_t capacity  = num_pairs * 2;
  cuco::static_map<key_pair, value_pair> map{capacity, sentinel_key, sentinel_value};

  thrust::device_vector<Key> insert_keys(num_pairs);
  thrust::device_vector<Value> insert_values(num_pairs);

  thrust::transform(thrust::device,
                    thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(num_pairs),
                    insert_keys.begin(),
                    [] __device__(auto i) {
                      return Key{i, i};
                    });

  thrust::transform(thrust::device,
                    thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(num_pairs),
                    insert_values.begin(),
                    [] __device__(auto i) {
                      return Value{i, i};
                    });

  auto insert_pairs =
    thrust::make_zip_iterator(thrust::make_tuple(insert_keys.begin(), insert_values.begin()));

  SECTION("All inserted keys-value pairs should be correctly recovered during find")
  {
    thrust::device_vector<Value> found_values(num_pairs);
    map.insert(insert_pairs, insert_pairs + num_pairs, hash_key_pair{}, key_pair_equals{});

    REQUIRE(num_pairs == map.get_size());

    map.find(insert_keys.begin(),
             insert_keys.end(),
             found_values.begin(),
             hash_key_pair{},
             key_pair_equals{});

    REQUIRE(thrust::equal(thrust::device,
                          insert_values.begin(),
                          insert_values.end(),
                          found_values.begin(),
                          [] __device__(value_pair lhs, value_pair rhs) {
                            return std::tie(lhs.f, lhs.s) == std::tie(rhs.f, rhs.s);
                          }));
  }
  SECTION("All inserted keys-value pairs should be contained")
  {
    thrust::device_vector<bool> contained(num_pairs);
    map.insert(insert_pairs, insert_pairs + num_pairs, hash_key_pair{}, key_pair_equals{});
    map.contains(insert_keys.begin(),
                 insert_keys.end(),
                 contained.begin(),
                 hash_key_pair{},
                 key_pair_equals{});
    REQUIRE(all_of(contained.begin(), contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("Non-inserted keys-value pairs should not be contained")
  {
    thrust::device_vector<bool> contained(num_pairs);
    map.contains(insert_keys.begin(),
                 insert_keys.end(),
                 contained.begin(),
                 hash_key_pair{},
                 key_pair_equals{});
    REQUIRE(
      none_of(contained.begin(), contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("Inserting unique keys should return insert success.")
  {
    auto m_view = map.get_device_mutable_view();
    REQUIRE(all_of(insert_pairs,
                   insert_pairs + num_pairs,
                   [m_view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                     return m_view.insert(pair, hash_key_pair{}, key_pair_equals{});
                   }));
  }

  SECTION("Cannot find any key in an empty hash map")
  {
    SECTION("non-const view")
    {
      auto view = map.get_device_view();
      REQUIRE(all_of(insert_pairs,
                     insert_pairs + num_pairs,
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       return view.find(pair.first, hash_key_pair{}, key_pair_equals{}) ==
                              view.end();
                     }));
    }

    SECTION("const view")
    {
      auto const view = map.get_device_view();
      REQUIRE(all_of(insert_pairs,
                     insert_pairs + num_pairs,
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) {
                       return view.find(pair.first, hash_key_pair{}, key_pair_equals{}) ==
                              view.end();
                     }));
    }
  }
}

TEMPLATE_TEST_CASE_SIG("Unique sequence of keys",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int64_t, dist_type::UNIQUE),
                       (int32_t, dist_type::UNIFORM),
                       (int64_t, dist_type::UNIFORM),
                       (int32_t, dist_type::GAUSSIAN),
                       (int64_t, dist_type::GAUSSIAN))
{
  using Key   = T;
  using Value = T;

  constexpr std::size_t num_keys{50'000'000};
  cuco::static_map<Key, Value> map{100'000'000, -1, -1};

  auto m_view = map.get_device_mutable_view();
  auto view   = map.get_device_view();

  std::vector<Key> h_keys(num_keys);
  std::vector<Value> h_values(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_keys<Dist, Key>(h_keys.begin(), h_keys.end());

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
    h_values[i]       = val;
  }

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<Value> d_values(h_values);
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);
  thrust::device_vector<Value> d_results(num_keys);
  thrust::device_vector<bool> d_contained(num_keys);

  // bulk function test cases
  SECTION("All inserted keys-value pairs should be correctly recovered during find")
  {
    map.insert(d_pairs.begin(), d_pairs.end());
    map.find(d_keys.begin(), d_keys.end(), d_results.begin());
    auto zip = thrust::make_zip_iterator(thrust::make_tuple(d_results.begin(), d_values.begin()));

    REQUIRE(all_of(zip, zip + num_keys, [] __device__(auto const& p) {
      return thrust::get<0>(p) == thrust::get<1>(p);
    }));
  }

  SECTION("All inserted keys-value pairs should be contained")
  {
    map.insert(d_pairs.begin(), d_pairs.end());
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(
      all_of(d_contained.begin(), d_contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("Non-inserted keys-value pairs should not be contained")
  {
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(
      none_of(d_contained.begin(), d_contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("Inserting unique keys should return insert success.")
  {
    if (Dist == dist_type::UNIQUE) {
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [m_view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       return m_view.insert(pair);
                     }));
    }
  }

  SECTION("Cannot find any key in an empty hash map with non-const view")
  {
    SECTION("non-const view")
    {
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       return view.find(pair.first) == view.end();
                     }));
    }
    SECTION("const view")
    {
      REQUIRE(all_of(
        d_pairs.begin(), d_pairs.end(), [view] __device__(cuco::pair_type<Key, Value> const& pair) {
          return view.find(pair.first) == view.end();
        }));
    }
  }

  SECTION("Keys are all found after inserting many keys.")
  {
    // Bulk insert keys
    thrust::for_each(thrust::device,
                     d_pairs.begin(),
                     d_pairs.end(),
                     [m_view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       m_view.insert(pair);
                     });

    SECTION("non-const view")
    {
      // All keys should be found
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       auto const found = view.find(pair.first);
                       return (found != view.end()) and (found->first.load() == pair.first and
                                                         found->second.load() == pair.second);
                     }));
    }
    SECTION("const view")
    {
      // All keys should be found
      REQUIRE(all_of(
        d_pairs.begin(), d_pairs.end(), [view] __device__(cuco::pair_type<Key, Value> const& pair) {
          auto const found = view.find(pair.first);
          return (found != view.end()) and
                 (found->first.load() == pair.first and found->second.load() == pair.second);
        }));
    }
  }
}

template <typename MapType, int CAPACITY>
__global__ void shared_memory_test_kernel(
  typename MapType::device_view const* const device_views,
  typename MapType::device_view::key_type const* const insterted_keys,
  typename MapType::device_view::mapped_type const* const inserted_values,
  const size_t number_of_elements,
  bool* const keys_exist,
  bool* const keys_and_values_correct)
{
  // Each block processes one map
  const size_t map_id = blockIdx.x;
  const size_t offset = map_id * number_of_elements;

  __shared__ typename MapType::pair_atomic_type sm_buffer[CAPACITY];

  auto g = cg::this_thread_block();
  typename MapType::device_view sm_device_view =
    MapType::device_view::make_copy(g, sm_buffer, device_views[map_id]);

  for (int i = g.thread_rank(); i < number_of_elements; i += g.size()) {
    auto found_pair_it = sm_device_view.find(insterted_keys[offset + i]);

    if (found_pair_it != sm_device_view.end()) {
      keys_exist[offset + i] = true;
      if (found_pair_it->first == insterted_keys[offset + i] and
          found_pair_it->second == inserted_values[offset + i]) {
        keys_and_values_correct[offset + i] = true;
      } else {
        keys_and_values_correct[offset + i] = false;
      }
    } else {
      keys_exist[offset + i]              = false;
      keys_and_values_correct[offset + i] = true;
    }
  }
}

TEMPLATE_TEST_CASE_SIG("Shared memory static map",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int64_t, dist_type::UNIQUE),
                       (int32_t, dist_type::UNIFORM),
                       (int64_t, dist_type::UNIFORM),
                       (int32_t, dist_type::GAUSSIAN),
                       (int64_t, dist_type::GAUSSIAN))
{
  using KeyType                = T;
  using ValueType              = T;
  using MapType                = cuco::static_map<KeyType, ValueType>;
  using DeviceViewType         = typename MapType::device_view;
  using DeviceViewIteratorType = typename DeviceViewType::iterator;

  constexpr std::size_t number_of_maps  = 1000;
  constexpr std::size_t elements_in_map = 500;
  constexpr std::size_t map_capacity    = 2 * elements_in_map;

  // one array for all maps, first elements_in_map element belong to map 0, second to map 1 and so
  // on
  std::vector<KeyType> h_keys(number_of_maps * elements_in_map);
  std::vector<ValueType> h_values(number_of_maps * elements_in_map);
  std::vector<cuco::pair_type<KeyType, ValueType>> h_pairs(number_of_maps * elements_in_map);

  // using std::unique_ptr because static_map does not have copy/move constructor/assignment
  // operator yet
  std::vector<std::unique_ptr<MapType>> maps;

  for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
    const std::size_t offset = map_id * elements_in_map;

    generate_keys<Dist, KeyType>(h_keys.begin() + offset,
                                 h_keys.begin() + offset + elements_in_map);

    for (std::size_t i = 0; i < elements_in_map; ++i) {
      KeyType key                = h_keys[offset + i];
      ValueType val              = key < std::numeric_limits<KeyType>::max() ? key + 1 : 0;
      h_values[offset + i]       = val;
      h_pairs[offset + i].first  = key;
      h_pairs[offset + i].second = val;
    }

    maps.push_back(std::make_unique<MapType>(map_capacity, -1, -1));
  }

  thrust::device_vector<KeyType> d_keys(h_keys);
  thrust::device_vector<ValueType> d_values(h_values);
  thrust::device_vector<cuco::pair_type<KeyType, ValueType>> d_pairs(h_pairs);

  SECTION("Keys are all found after insertion.")
  {
    std::vector<DeviceViewType> h_device_views;
    for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
      const std::size_t offset = map_id * elements_in_map;

      MapType* map = maps[map_id].get();
      map->insert(d_pairs.begin() + offset, d_pairs.begin() + offset + elements_in_map);
      h_device_views.push_back(map->get_device_view());
    }
    thrust::device_vector<DeviceViewType> d_device_views(h_device_views);

    thrust::device_vector<bool> d_keys_exist(number_of_maps * elements_in_map);
    thrust::device_vector<bool> d_keys_and_values_correct(number_of_maps * elements_in_map);

    shared_memory_test_kernel<MapType, map_capacity>
      <<<number_of_maps, 64>>>(d_device_views.data().get(),
                               d_keys.data().get(),
                               d_values.data().get(),
                               elements_in_map,
                               d_keys_exist.data().get(),
                               d_keys_and_values_correct.data().get());

    REQUIRE(d_keys_exist.size() == d_keys_and_values_correct.size());
    auto zip = thrust::make_zip_iterator(
      thrust::make_tuple(d_keys_exist.begin(), d_keys_and_values_correct.begin()));

    REQUIRE(all_of(zip, zip + d_keys_exist.size(), [] __device__(auto const& z) {
      return thrust::get<0>(z) and thrust::get<1>(z);
    }));
  }

  SECTION("No key is found before insertion.")
  {
    std::vector<DeviceViewType> h_device_views;
    for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
      h_device_views.push_back(maps[map_id].get()->get_device_view());
    }
    thrust::device_vector<DeviceViewType> d_device_views(h_device_views);

    thrust::device_vector<bool> d_keys_exist(number_of_maps * elements_in_map);
    thrust::device_vector<bool> d_keys_and_values_correct(number_of_maps * elements_in_map);

    shared_memory_test_kernel<MapType, map_capacity>
      <<<number_of_maps, 64>>>(d_device_views.data().get(),
                               d_keys.data().get(),
                               d_values.data().get(),
                               elements_in_map,
                               d_keys_exist.data().get(),
                               d_keys_and_values_correct.data().get());

    REQUIRE(none_of(d_keys_exist.begin(), d_keys_exist.end(), [] __device__(const bool key_found) {
      return key_found;
    }));
  }
}

template <typename K, typename V, std::size_t N>
__global__ void shared_memory_hash_table_kernel(bool* key_found)
{
  namespace cg   = cooperative_groups;
  using map_type = typename cuco::static_map<K, V, cuda::thread_scope_block>::device_mutable_view;
  using find_map_type = typename cuco::static_map<K, V, cuda::thread_scope_block>::device_view;
  __shared__ typename map_type::slot_type slots[N];
  auto map = map_type::make_from_uninitialized_slots(cg::this_thread_block(), &slots[0], N, -1, -1);

  auto g            = cg::this_thread_block();
  std::size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  int rank          = g.thread_rank();

  // insert {thread_rank, thread_rank} for each thread in thread-block
  map.insert(cuco::pair<int, int>(rank, rank));
  g.sync();

  auto find_map       = find_map_type(map);
  auto retrieved_pair = find_map.find(rank);
  if (retrieved_pair != find_map.end() && retrieved_pair->second == rank) {
    key_found[index] = true;
  }
}

TEMPLATE_TEST_CASE("Shared memory slots.", "", int32_t)
{
  constexpr std::size_t N = 256;
  thrust::device_vector<bool> key_found(N, false);
  shared_memory_hash_table_kernel<TestType, TestType, N><<<8, 32>>>(key_found.data().get());

  REQUIRE(all_of(key_found.begin(), key_found.end(), thrust::identity<bool>{}));
}
