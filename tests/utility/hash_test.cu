#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <test_utils.hpp>

#include <cuco/detail/__config>
#include <cuco/hash_functions.cuh>

#include <thrust/device_vector.h>

#include <catch2/catch_template_test_macros.hpp>
#include <catch2/catch_test_macros.hpp>

#include <cstddef>

template <int32_t Words>
struct large_key {
  constexpr __host__ __device__ large_key(int32_t value) noexcept
  {
    for (int32_t i = 0; i < Words; ++i) {
      data_[i] = value;
    }
  }

 private:
  int32_t data_[Words];
};

template <typename Hash, typename... HashConstructorArgs>
static __host__ __device__ bool check_hash_result(
  typename Hash::argument_type const& key,
  typename Hash::result_type expected,
  HashConstructorArgs&&... hash_constructor_args) noexcept
{
  Hash h(std::forward<HashConstructorArgs>(hash_constructor_args)...);
  return (h(key) == expected);
}

// Overload for hash functions without a seed
template <typename Hash>
__host__ __device__ bool check_hash_result(typename Hash::argument_type const& key,
                                           typename Hash::result_type expected) noexcept
{
  Hash h;
  return (h(key) == expected);
}

template <typename OutputIter>
__global__ void check_identity_hash_result_kernel(OutputIter result)
{
  int i = 0;

  result[i++] = check_hash_result<cuco::identity_hash<signed char>>(0, 0);
  result[i++] = check_hash_result<cuco::identity_hash<signed char>>(
    std::numeric_limits<signed char>::max(), std::numeric_limits<signed char>::max());

  result[i++] = check_hash_result<cuco::identity_hash<int32_t>>(0, 0);
  result[i++] = check_hash_result<cuco::identity_hash<int32_t>>(
    std::numeric_limits<int32_t>::max(), std::numeric_limits<int32_t>::max());

  result[i++] = check_hash_result<cuco::identity_hash<int64_t>>(0, 0);
  result[i++] = check_hash_result<cuco::identity_hash<int64_t>>(
    std::numeric_limits<int64_t>::max(), std::numeric_limits<int64_t>::max());
}

TEST_CASE("Test cuco::identity_hash", "")
{
  SECTION("Check if host-generated hash values match the identity function.")
  {
    CHECK(check_hash_result<cuco::identity_hash<signed char>>(0, 0));
    CHECK(check_hash_result<cuco::identity_hash<signed char>>(
      std::numeric_limits<signed char>::max(), std::numeric_limits<signed char>::max()));

    CHECK(check_hash_result<cuco::identity_hash<int32_t>>(0, 0));
    CHECK(check_hash_result<cuco::identity_hash<int32_t>>(std::numeric_limits<int32_t>::max(),
                                                          std::numeric_limits<int32_t>::max()));

    CHECK(check_hash_result<cuco::identity_hash<int64_t>>(0, 0));
    CHECK(check_hash_result<cuco::identity_hash<int64_t>>(std::numeric_limits<int64_t>::max(),
                                                          std::numeric_limits<int64_t>::max()));
  }
  SECTION("Check if device-generated hash values match the identity function.")
  {
    thrust::device_vector<bool> result(7, true);

    check_identity_hash_result_kernel<<<1, 1>>>(result.begin());

    CHECK(cuco::test::all_of(result.begin(), result.end(), [] __device__(bool v) { return v; }));
  }
}

template <typename OutputIter>
__global__ void check_hash_result_kernel_64(OutputIter result)
{
  int i = 0;

  result[i++] = check_hash_result<cuco::xxhash_64<char>>(0, 16804241149081757544, 0);
  result[i++] = check_hash_result<cuco::xxhash_64<char>>(42, 765293966243412708, 0);
  result[i++] = check_hash_result<cuco::xxhash_64<char>>(0, 9486749600008296231, 42);

  result[i++] = check_hash_result<cuco::xxhash_64<int32_t>>(0, 4246796580750024372, 0);
  result[i++] = check_hash_result<cuco::xxhash_64<int32_t>>(0, 3614696996920510707, 42);
  result[i++] = check_hash_result<cuco::xxhash_64<int32_t>>(42, 15516826743637085169, 0);
  result[i++] = check_hash_result<cuco::xxhash_64<int32_t>>(123456789, 9462334144942111946, 0);

  result[i++] = check_hash_result<cuco::xxhash_64<int64_t>>(0, 3803688792395291579, 0);
  result[i++] = check_hash_result<cuco::xxhash_64<int64_t>>(0, 13194218611613725804, 42);
  result[i++] = check_hash_result<cuco::xxhash_64<int64_t>>(42, 13066772586158965587, 0);
  result[i++] = check_hash_result<cuco::xxhash_64<int64_t>>(123456789, 14662639848940634189, 0);

#if defined(CUCO_HAS_INT128)
  result[i++] = check_hash_result<cuco::xxhash_64<__int128>>(123456789, 7986913354431084250, 0);
#endif

  result[i++] =
    check_hash_result<cuco::xxhash_64<large_key<32>>>(123456789, 2031761887105658523, 0);
}

TEST_CASE("Test cuco::xxhash_64", "")
{
  // Reference hash values were computed using https://github.com/Cyan4973/xxHash
  SECTION("Check if host-generated hash values match the reference implementation.")
  {
    CHECK(check_hash_result<cuco::xxhash_64<char>>(0, 16804241149081757544, 0));
    CHECK(check_hash_result<cuco::xxhash_64<char>>(42, 765293966243412708, 0));
    CHECK(check_hash_result<cuco::xxhash_64<char>>(0, 9486749600008296231, 42));

    CHECK(check_hash_result<cuco::xxhash_64<int32_t>>(0, 4246796580750024372, 0));
    CHECK(check_hash_result<cuco::xxhash_64<int32_t>>(0, 3614696996920510707, 42));
    CHECK(check_hash_result<cuco::xxhash_64<int32_t>>(42, 15516826743637085169, 0));
    CHECK(check_hash_result<cuco::xxhash_64<int32_t>>(123456789, 9462334144942111946, 0));

    CHECK(check_hash_result<cuco::xxhash_64<int64_t>>(0, 3803688792395291579, 0));
    CHECK(check_hash_result<cuco::xxhash_64<int64_t>>(0, 13194218611613725804, 42));
    CHECK(check_hash_result<cuco::xxhash_64<int64_t>>(42, 13066772586158965587, 0));
    CHECK(check_hash_result<cuco::xxhash_64<int64_t>>(123456789, 14662639848940634189, 0));

#if defined(CUCO_HAS_INT128)
    CHECK(check_hash_result<cuco::xxhash_64<__int128>>(123456789, 7986913354431084250, 0));
#endif

    // 32*4=128-byte key to test the pipelined outermost hashing loop
    CHECK(check_hash_result<cuco::xxhash_64<large_key<32>>>(123456789, 2031761887105658523, 0));
  }

  SECTION("Check if device-generated hash values match the reference implementation.")
  {
    thrust::device_vector<bool> result(10);

    check_hash_result_kernel_64<<<1, 1>>>(result.begin());

    CHECK(cuco::test::all_of(result.begin(), result.end(), thrust::identity<bool>{}));
  }
}

template <typename OutputIter>
__global__ void check_hash_result_kernel_32(OutputIter result)
{
  int i = 0;

  result[i++] = check_hash_result<cuco::xxhash_32<char>>(0, 3479547966, 0);
  result[i++] = check_hash_result<cuco::xxhash_32<char>>(42, 3774771295, 0);
  result[i++] = check_hash_result<cuco::xxhash_32<char>>(0, 2099223482, 42);

  result[i++] = check_hash_result<cuco::xxhash_32<int32_t>>(0, 148298089, 0);
  result[i++] = check_hash_result<cuco::xxhash_32<int32_t>>(0, 2132181312, 42);
  result[i++] = check_hash_result<cuco::xxhash_32<int32_t>>(42, 1161967057, 0);
  result[i++] = check_hash_result<cuco::xxhash_32<int32_t>>(123456789, 2987034094, 0);

  result[i++] = check_hash_result<cuco::xxhash_32<int64_t>>(0, 3736311059, 0);
  result[i++] = check_hash_result<cuco::xxhash_32<int64_t>>(0, 1076387279, 42);
  result[i++] = check_hash_result<cuco::xxhash_32<int64_t>>(42, 2332451213, 0);
  result[i++] = check_hash_result<cuco::xxhash_32<int64_t>>(123456789, 1561711919, 0);

#if defined(CUCO_HAS_INT128)
  result[i++] = check_hash_result<cuco::xxhash_32<__int128>>(123456789, 1846633701, 0);
#endif

  result[i++] = check_hash_result<cuco::xxhash_32<large_key<32>>>(123456789, 3715432378, 0);
}

TEST_CASE("Test cuco::xxhash_32", "")
{
  // Reference hash values were computed using https://github.com/Cyan4973/xxHash
  SECTION("Check if host-generated hash values match the reference implementation.")
  {
    CHECK(check_hash_result<cuco::xxhash_32<char>>(0, 3479547966, 0));
    CHECK(check_hash_result<cuco::xxhash_32<char>>(42, 3774771295, 0));
    CHECK(check_hash_result<cuco::xxhash_32<char>>(0, 2099223482, 42));

    CHECK(check_hash_result<cuco::xxhash_32<int32_t>>(0, 148298089, 0));
    CHECK(check_hash_result<cuco::xxhash_32<int32_t>>(0, 2132181312, 42));
    CHECK(check_hash_result<cuco::xxhash_32<int32_t>>(42, 1161967057, 0));
    CHECK(check_hash_result<cuco::xxhash_32<int32_t>>(123456789, 2987034094, 0));

    CHECK(check_hash_result<cuco::xxhash_32<int64_t>>(0, 3736311059, 0));
    CHECK(check_hash_result<cuco::xxhash_32<int64_t>>(0, 1076387279, 42));
    CHECK(check_hash_result<cuco::xxhash_32<int64_t>>(42, 2332451213, 0));
    CHECK(check_hash_result<cuco::xxhash_32<int64_t>>(123456789, 1561711919, 0));

#if defined(CUCO_HAS_INT128)
    CHECK(check_hash_result<cuco::xxhash_32<__int128>>(123456789, 1846633701, 0));
#endif

    // 32*4=128-byte key to test the pipelined outermost hashing loop
    CHECK(check_hash_result<cuco::xxhash_32<large_key<32>>>(123456789, 3715432378, 0));
  }

  SECTION("Check if device-generated hash values match the reference implementation.")
  {
    thrust::device_vector<bool> result(20, true);

    check_hash_result_kernel_32<<<1, 1>>>(result.begin());

    CHECK(cuco::test::all_of(result.begin(), result.end(), thrust::identity<bool>{}));
  }
}

TEMPLATE_TEST_CASE_SIG("Static vs. dynamic key hash test",
                       "",
                       ((typename Hash), Hash),
                       (cuco::murmurhash3_32<char>),
                       (cuco::murmurhash3_32<int32_t>),
                       (cuco::xxhash_32<char>),
                       (cuco::xxhash_32<int32_t>),
                       (cuco::xxhash_64<char>),
                       (cuco::xxhash_64<int32_t>))
{
  using key_type = typename Hash::argument_type;

  Hash hash;
  key_type key = 42;

  SECTION("Identical keys with static and dynamic key size should have the same hash value.")
  {
    CHECK(hash(key) ==
          hash.compute_hash(reinterpret_cast<std::byte const*>(&key), sizeof(key_type)));
  }
}

template <typename OutputIter>
__global__ void check_murmurhash3_128_result_kernel(OutputIter result)
{
  int i = 0;

  result[i++] = check_hash_result<cuco::murmurhash3_x64_128<int32_t>, uint64_t>(
    0, {14961230494313510588ull, 6383328099726337777ull}, 0);
  result[i++] = check_hash_result<cuco::murmurhash3_x64_128<int32_t>, uint64_t>(
    9, {1779292183511753683ull, 16298496441448380334ull}, 0);
  result[i++] = check_hash_result<cuco::murmurhash3_x64_128<int32_t>, uint64_t>(
    42, {2913627637088662735ull, 16344193523890567190ull}, 0);
  result[i++] = check_hash_result<cuco::murmurhash3_x64_128<int32_t>, uint64_t>(
    42, {2248879576374326886ull, 18006515275339376488ull}, 42);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int32_t, 2>>, uint64_t>(
      {2, 2}, {12221386834995143465ull, 6690950894782946573ull}, 0);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int32_t, 3>>, uint64_t>(
      {1, 4, 9}, {299140022350411792ull, 9891903873182035274ull}, 42);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int32_t, 4>>, uint64_t>(
      {42, 64, 108, 1024}, {4333511168876981289ull, 4659486988434316416ull}, 63);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int32_t, 16>>, uint64_t>(
      {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16},
      {3302412811061286680ull, 7070355726356610672ull},
      1024);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int64_t, 2>>, uint64_t>(
      {2, 2}, {8554944597931919519ull, 14938998000509429729ull}, 0);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int64_t, 3>>, uint64_t>(
      {1, 4, 9}, {13442629947720186435ull, 7061727494178573325ull}, 42);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int64_t, 4>>, uint64_t>(
      {42, 64, 108, 1024}, {8786399719555989948ull, 14954183901757012458ull}, 63);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int64_t, 16>>, uint64_t>(
      {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16},
      {15409921801541329777ull, 10546487400963404004ull},
      1024);

  result[i++] = check_hash_result<cuco::murmurhash3_x86_128<int32_t>, uint32_t>(
    0, {3422973727, 2656139328, 2656139328, 2656139328}, 0);
  result[i++] = check_hash_result<cuco::murmurhash3_x86_128<int32_t>, uint32_t>(
    9, {2808089785, 314604614, 314604614, 314604614}, 0);
  result[i++] = check_hash_result<cuco::murmurhash3_x86_128<int32_t>, uint32_t>(
    42, {3611919118, 1962256489, 1962256489, 1962256489}, 0);
  result[i++] = check_hash_result<cuco::murmurhash3_x86_128<int32_t>, uint32_t>(
    42, {3399017053, 732469929, 732469929, 732469929}, 42);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int32_t, 2>>, uint32_t>(
      {2, 2}, {1234494082, 1431451587, 431049201, 431049201}, 0);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int32_t, 3>>, uint32_t>(
      {1, 4, 9}, {2516796247, 2757675829, 778406919, 2453259553}, 42);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int32_t, 4>>, uint32_t>(
      {42, 64, 108, 1024}, {2686265656, 591236665, 3797082165, 2731908938}, 63);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int32_t, 16>>, uint32_t>(
      {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16},
      {3918256832, 4205523739, 1707810111, 1625952473},
      1024);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int64_t, 2>>, uint32_t>(
      {2, 2}, {3811075945, 727160712, 3510740342, 235225510}, 0);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int64_t, 3>>, uint32_t>(
      {1, 4, 9}, {2817194959, 206796677, 3391242768, 248681098}, 42);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int64_t, 4>>, uint32_t>(
      {42, 64, 108, 1024}, {2335912146, 1566515912, 760710030, 452077451}, 63);
  result[i++] =
    check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int64_t, 16>>, uint32_t>(
      {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16},
      {1101169764, 1758958147, 2406511780, 2903571412},
      1024);
}

TEST_CASE("Test cuco::murmurhash3_x64_128", "")
{
  // Reference hash values were computed using
  // https://github.com/aappleby/smhasher/blob/master/src/MurmurHash3.cpp

  SECTION("Check if host-generated hash values match the reference implementation.")
  {
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<int32_t>, uint64_t>(
      0, {14961230494313510588ull, 6383328099726337777ull}, 0));
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<int32_t>, uint64_t>(
      9, {1779292183511753683ull, 16298496441448380334ull}, 0));
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<int32_t>, uint64_t>(
      42, {2913627637088662735ull, 16344193523890567190ull}, 0));
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<int32_t>, uint64_t>(
      42, {2248879576374326886ull, 18006515275339376488ull}, 42));
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int32_t, 2>>, uint64_t>(
      {2, 2}, {12221386834995143465ull, 6690950894782946573ull}, 0));
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int32_t, 3>>, uint64_t>(
      {1, 4, 9}, {299140022350411792ull, 9891903873182035274ull}, 42));
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int32_t, 4>>, uint64_t>(
      {42, 64, 108, 1024}, {4333511168876981289ull, 4659486988434316416ull}, 63));
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int32_t, 16>>, uint64_t>(
      {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16},
      {3302412811061286680ull, 7070355726356610672ull},
      1024));
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int64_t, 2>>, uint64_t>(
      {2, 2}, {8554944597931919519ull, 14938998000509429729ull}, 0));
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int64_t, 3>>, uint64_t>(
      {1, 4, 9}, {13442629947720186435ull, 7061727494178573325ull}, 42));
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int64_t, 4>>, uint64_t>(
      {42, 64, 108, 1024}, {8786399719555989948ull, 14954183901757012458ull}, 63));
    CHECK(check_hash_result<cuco::murmurhash3_x64_128<cuda::std::array<int64_t, 16>>, uint64_t>(
      {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16},
      {15409921801541329777ull, 10546487400963404004ull},
      1024));

    CHECK(check_hash_result<cuco::murmurhash3_x86_128<int32_t>, uint32_t>(
      0, {3422973727, 2656139328, 2656139328, 2656139328}, 0));
    CHECK(check_hash_result<cuco::murmurhash3_x86_128<int32_t>, uint32_t>(
      9, {2808089785, 314604614, 314604614, 314604614}, 0));
    CHECK(check_hash_result<cuco::murmurhash3_x86_128<int32_t>, uint32_t>(
      42, {3611919118, 1962256489, 1962256489, 1962256489}, 0));
    CHECK(check_hash_result<cuco::murmurhash3_x86_128<int32_t>, uint32_t>(
      42, {3399017053, 732469929, 732469929, 732469929}, 42));
    CHECK(check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int32_t, 2>>, uint32_t>(
      {2, 2}, {1234494082, 1431451587, 431049201, 431049201}, 0));
    CHECK(check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int32_t, 3>>, uint32_t>(
      {1, 4, 9}, {2516796247, 2757675829, 778406919, 2453259553}, 42));
    CHECK(check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int32_t, 4>>, uint32_t>(
      {42, 64, 108, 1024}, {2686265656, 591236665, 3797082165, 2731908938}, 63));
    CHECK(check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int32_t, 16>>, uint32_t>(
      {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16},
      {3918256832, 4205523739, 1707810111, 1625952473},
      1024));
    CHECK(check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int64_t, 2>>, uint32_t>(
      {2, 2}, {3811075945, 727160712, 3510740342, 235225510}, 0));
    CHECK(check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int64_t, 3>>, uint32_t>(
      {1, 4, 9}, {2817194959, 206796677, 3391242768, 248681098}, 42));
    CHECK(check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int64_t, 4>>, uint32_t>(
      {42, 64, 108, 1024}, {2335912146, 1566515912, 760710030, 452077451}, 63));
    CHECK(check_hash_result<cuco::murmurhash3_x86_128<cuda::std::array<int64_t, 16>>, uint32_t>(
      {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16},
      {1101169764, 1758958147, 2406511780, 2903571412},
      1024));
  }

  SECTION("Check if device-generated hash values match the reference implementation.")
  {
    thrust::device_vector<bool> result(24, true);

    check_murmurhash3_128_result_kernel<<<1, 1>>>(result.begin());

    CHECK(cuco::test::all_of(result.begin(), result.end(), thrust::identity<bool>{}));
  }
}
