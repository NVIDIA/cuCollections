#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <algorithm>
#include <catch2/catch.hpp>
#include <cuco/legacy_static_map.cuh>

namespace {
// Thrust logical algorithms (any_of/all_of/none_of) don't work with device
// lambdas: See https://github.com/thrust/thrust/issues/1062
template <typename Iterator, typename Predicate>
bool all_of(Iterator begin, Iterator end, Predicate p)
{
  auto size = thrust::distance(begin, end);
  return size == thrust::count_if(begin, end, p);
}

template <typename Iterator, typename Predicate>
bool any_of(Iterator begin, Iterator end, Predicate p)
{
  return thrust::count_if(begin, end, p) > 0;
}

template <typename Iterator, typename Predicate>
bool none_of(Iterator begin, Iterator end, Predicate p)
{
  return not all_of(begin, end, p);
}
}  // namespace

enum class dist_type { UNIQUE, UNIFORM, GAUSSIAN };

template <dist_type Dist, typename Key, typename OutputIt>
static void generate_keys(OutputIt output_begin, OutputIt output_end)
{
  auto num_keys = std::distance(output_begin, output_end);

  std::random_device rd;
  std::mt19937 gen{rd()};

  switch (Dist) {
    case dist_type::UNIQUE:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = i;
      }
      break;
    case dist_type::UNIFORM:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(gen()));
      }
      break;
    case dist_type::GAUSSIAN:
      std::normal_distribution<> dg{1e9, 1e7};
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(dg(gen)));
      }
      break;
  }
}

TEMPLATE_TEST_CASE_SIG("Unique sequence of keys",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int32_t, dist_type::UNIFORM),
                       (int32_t, dist_type::GAUSSIAN))
{
  using Key   = T;
  using Value = T;

  constexpr std::size_t num_keys{50'000'000};
  cuco::legacy_static_map<Key, Value> map{100'000'000, -1, -1};

  auto m_view = map.get_device_mutable_view();
  auto view   = map.get_device_view();

  std::vector<Key> h_keys(num_keys);
  std::vector<Value> h_values(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);
  std::vector<Value> h_results(num_keys);
  std::vector<bool> h_contained(num_keys);

  generate_keys<Dist, Key>(h_keys.begin(), h_keys.end());

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
    h_values[i]       = val;
  }

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<Value> d_values(h_values);
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);
  thrust::device_vector<Value> d_results(num_keys);
  thrust::device_vector<bool> d_contained(num_keys);

  // bulk function test cases
  SECTION("All inserted keys-value pairs should be correctly recovered during find")
  {
    map.insert(d_pairs.begin(), d_pairs.end());
    map.find(d_keys.begin(), d_keys.end(), d_results.begin());
    auto zip = thrust::make_zip_iterator(thrust::make_tuple(d_results.begin(), d_values.begin()));

    REQUIRE(all_of(zip, zip + num_keys, [] __device__(auto const& p) {
      return thrust::get<0>(p) == thrust::get<1>(p);
    }));
  }

  SECTION("All inserted keys-value pairs should be contained")
  {
    map.insert(d_pairs.begin(), d_pairs.end());
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(
      all_of(d_contained.begin(), d_contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("Non-inserted keys-value pairs should not be contained")
  {
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(
      none_of(d_contained.begin(), d_contained.end(), [] __device__(bool const& b) { return b; }));
  }

  
  SECTION("Inserting unique keys should return insert success.")
  {
    if (Dist == dist_type::UNIQUE) {
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [m_view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       return m_view.insert(pair);
                     }));
    }
  }

  SECTION("Cannot find any key in an empty hash map with non-const view")
  {
    SECTION("non-const view")
    {
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       return view.find(pair.first) == view.end();
                     }));
    }
    SECTION("const view")
    {
      REQUIRE(all_of(
        d_pairs.begin(), d_pairs.end(), [view] __device__(cuco::pair_type<Key, Value> const& pair) {
          return view.find(pair.first) == view.end();
        }));
    }
  }

  SECTION("Keys are all found after inserting many keys.")
  {
    // Bulk insert keys
    thrust::for_each(thrust::device,
                     d_pairs.begin(),
                     d_pairs.end(),
                     [m_view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       m_view.insert(pair);
                     });
    hipDeviceSynchronize();
    
    SECTION("non-const view")
    {
      // All keys should be found
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       auto const found = view.find(pair.first);
                       return (found != view.end()) and (found->first == pair.first and
                                                         found->second == pair.second);
                     }));
    }
    SECTION("const view")
    {
      // All keys should be found
      REQUIRE(all_of(
        d_pairs.begin(), d_pairs.end(), [view] __device__(cuco::pair_type<Key, Value> const& pair) {
          auto const found = view.find(pair.first);
          return (found != view.end()) and
                 (found->first == pair.first and found->second == pair.second);
        }));
    }
  }
}