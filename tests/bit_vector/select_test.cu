#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utils.hpp>

#include <cuco/detail/trie/bit_vector/bit_vector.cuh>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>

#include <catch2/catch_test_macros.hpp>

template <class BitVectorRef, typename size_type, typename OutputIt>
__global__ void select0_kernel(BitVectorRef ref, size_type num_elements, OutputIt output)
{
  size_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = gridDim.x * blockDim.x;
  while (index < num_elements) {
    output[index] = ref.select0(index);
    index += stride;
  }
}

extern bool modulo_bitgen(uint64_t i);  // Defined in get_test.cu

TEST_CASE("Select test", "")
{
  cuco::experimental::detail::bit_vector bv;

  using size_type = cuco::experimental::detail::bit_vector::size_type;
  constexpr size_type num_elements{4000};

  size_type num_set = 0;
  for (size_type i = 0; i < num_elements; i++) {
    bv.append(modulo_bitgen(i));
    num_set += modulo_bitgen(i);
  }
  bv.build();
  auto ref = bv.ref(cuco::experimental::bv_read);

  // Check select
  {
    thrust::device_vector<size_type> keys(num_set);
    thrust::sequence(keys.begin(), keys.end(), 0);

    thrust::device_vector<size_type> d_selects(num_set);

    bv.selects(keys.begin(), keys.end(), d_selects.begin());

    thrust::host_vector<size_type> h_selects = d_selects;

    size_type num_matches = 0;
    size_type cur_set_pos = -1lu;
    for (size_type i = 0; i < num_set; i++) {
      do {
        cur_set_pos++;
      } while (cur_set_pos < num_elements and !modulo_bitgen(cur_set_pos));

      num_matches += cur_set_pos == h_selects[i];
    }
    REQUIRE(num_matches == num_set);
  }

  // Check select0
  {
    size_type num_not_set = num_elements - num_set;

    thrust::device_vector<size_type> device_result(num_not_set);
    select0_kernel<<<1, 1024>>>(ref, num_not_set, device_result.data());
    thrust::host_vector<size_type> host_result = device_result;

    size_type num_matches     = 0;
    size_type cur_not_set_pos = -1lu;
    for (size_type i = 0; i < num_not_set; i++) {
      do {
        cur_not_set_pos++;
      } while (cur_not_set_pos < num_elements and modulo_bitgen(cur_not_set_pos));

      num_matches += cur_not_set_pos == host_result[i];
    }
    REQUIRE(num_matches == num_not_set);
  }
}
