#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utils.hpp>

#include <cuco/bit_vector.cuh>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <catch2/catch_test_macros.hpp>

template <class BitVectorRef>
__global__ void get_kernel(BitVectorRef ref, size_t n, uint32_t* output)
{
  size_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = gridDim.x * blockDim.x;
  while (index < n) {
    output[index] = ref.get(index);
    index += stride;
  }
}

bool modulo_bitgen(uint32_t i) { return i % 7 == 0; }

TEST_CASE("Get test", "")
{
  constexpr std::size_t num_elements{400};

  using Key = uint64_t;
  cuco::experimental::bit_vector bv{cuco::experimental::extent<std::size_t>{400}};

  uint32_t num_set_ref = 0;
  for (size_t i = 0; i < num_elements; i++) {
    bv.add(modulo_bitgen(i));
    num_set_ref += modulo_bitgen(i);
  }
  bv.build();

  auto ref = bv.ref(cuco::experimental::bv_read);
  thrust::device_vector<uint32_t> get_result(num_elements);
  get_kernel<<<1, 1024>>>(ref, num_elements, thrust::raw_pointer_cast(get_result.data()));

  size_t num_set = thrust::reduce(thrust::device, get_result.begin(), get_result.end(), 0);
  REQUIRE(num_set == num_set_ref);
}
