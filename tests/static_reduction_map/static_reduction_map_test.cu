#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <algorithm>
#include <catch2/catch.hpp>
#include <cuco/static_reduction_map.cuh>
#include <limits>

namespace {
namespace cg = cooperative_groups;

// Thrust logical algorithms (any_of/all_of/none_of) don't work with device
// lambdas: See https://github.com/thrust/thrust/issues/1062
template <typename Iterator, typename Predicate>
bool all_of(Iterator begin, Iterator end, Predicate p)
{
  auto size = thrust::distance(begin, end);
  return size == thrust::count_if(begin, end, p);
}

template <typename Iterator, typename Predicate>
bool any_of(Iterator begin, Iterator end, Predicate p)
{
  return thrust::count_if(begin, end, p) > 0;
}

template <typename Iterator, typename Predicate>
bool none_of(Iterator begin, Iterator end, Predicate p)
{
  return not all_of(begin, end, p);
}
}  // namespace

enum class dist_type { UNIQUE, UNIFORM, GAUSSIAN };

template <dist_type Dist, typename Key, typename OutputIt>
static void generate_keys(OutputIt output_begin, OutputIt output_end)
{
  auto num_keys = std::distance(output_begin, output_end);

  std::random_device rd;
  std::mt19937 gen{rd()};

  switch (Dist) {
    case dist_type::UNIQUE:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = i;
      }
      break;
    case dist_type::UNIFORM:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(gen()));
      }
      break;
    case dist_type::GAUSSIAN:
      std::normal_distribution<> dg{1e9, 1e7};
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(dg(gen)));
      }
      break;
  }
}

TEMPLATE_TEST_CASE_SIG("Unique sequence of keys",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int64_t, dist_type::UNIQUE),
                       (int32_t, dist_type::UNIFORM),
                       (int64_t, dist_type::UNIFORM),
                       (int32_t, dist_type::GAUSSIAN),
                       (int64_t, dist_type::GAUSSIAN))
{
  using Key   = T;
  using Value = T;

  constexpr std::size_t num_slots{50'000'000};
  cuco::static_reduction_map<cuco::reduce_add<Value>, Key, Value> map{num_slots, -1};

}
