#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <algorithm>
#include <catch2/catch.hpp>
#include <cuco/static_reduction_map.cuh>
#include <limits>

namespace {
namespace cg = cooperative_groups;

// Thrust logical algorithms (any_of/all_of/none_of) don't work with device
// lambdas: See https://github.com/thrust/thrust/issues/1062
template <typename Iterator, typename Predicate>
bool all_of(Iterator begin, Iterator end, Predicate p)
{
  auto size = thrust::distance(begin, end);
  return size == thrust::count_if(begin, end, p);
}

template <typename Iterator, typename Predicate>
bool any_of(Iterator begin, Iterator end, Predicate p)
{
  return thrust::count_if(begin, end, p) > 0;
}

template <typename Iterator, typename Predicate>
bool none_of(Iterator begin, Iterator end, Predicate p)
{
  return not all_of(begin, end, p);
}
}  // namespace

enum class dist_type { UNIQUE, UNIFORM, GAUSSIAN };

template <dist_type Dist, typename Key, typename OutputIt>
static void generate_keys(OutputIt output_begin, OutputIt output_end)
{
  auto num_keys = std::distance(output_begin, output_end);

  std::random_device rd;
  std::mt19937 gen{rd()};

  switch (Dist) {
    case dist_type::UNIQUE:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = i;
      }
      break;
    case dist_type::UNIFORM:
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(gen()));
      }
      break;
    case dist_type::GAUSSIAN:
      std::normal_distribution<> dg{1e9, 1e7};
      for (auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(dg(gen)));
      }
      break;
  }
}

TEMPLATE_TEST_CASE_SIG("Unique sequence of keys",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int64_t, dist_type::UNIQUE),
                       (int32_t, dist_type::UNIFORM),
                       (int64_t, dist_type::UNIFORM),
                       (int32_t, dist_type::GAUSSIAN),
                       (int64_t, dist_type::GAUSSIAN))
{
  using Key   = T;
  using Value = T;

  constexpr std::size_t num_keys{50'000'000};
  cuco::static_reduction_map<Key, Value> map{100'000'000, -1, -1};

  auto m_view = map.get_device_mutable_view();
  auto view   = map.get_device_view();

  std::vector<Key> h_keys(num_keys);
  std::vector<Value> h_values(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_keys<Dist, Key>(h_keys.begin(), h_keys.end());

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
    h_values[i]       = val;
  }

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<Value> d_values(h_values);
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);
  thrust::device_vector<Value> d_results(num_keys);
  thrust::device_vector<bool> d_contained(num_keys);

  // bulk function test cases
  SECTION("All inserted keys-value pairs should be correctly recovered during find")
  {
    map.insert(d_pairs.begin(), d_pairs.end());
    map.find(d_keys.begin(), d_keys.end(), d_results.begin());
    auto zip = thrust::make_zip_iterator(thrust::make_tuple(d_results.begin(), d_values.begin()));

    REQUIRE(all_of(zip, zip + num_keys, [] __device__(auto const& p) {
      return thrust::get<0>(p) == thrust::get<1>(p);
    }));
  }

  SECTION("All inserted keys-value pairs should be contained")
  {
    map.insert(d_pairs.begin(), d_pairs.end());
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(
      all_of(d_contained.begin(), d_contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("Non-inserted keys-value pairs should not be contained")
  {
    map.contains(d_keys.begin(), d_keys.end(), d_contained.begin());

    REQUIRE(
      none_of(d_contained.begin(), d_contained.end(), [] __device__(bool const& b) { return b; }));
  }

  SECTION("Inserting unique keys should return insert success.")
  {
    if (Dist == dist_type::UNIQUE) {
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [m_view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       return m_view.insert(pair);
                     }));
    }
  }

  SECTION("Cannot find any key in an empty hash map with non-const view")
  {
    SECTION("non-const view")
    {
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       return view.find(pair.first) == view.end();
                     }));
    }
    SECTION("const view")
    {
      REQUIRE(all_of(
        d_pairs.begin(), d_pairs.end(), [view] __device__(cuco::pair_type<Key, Value> const& pair) {
          return view.find(pair.first) == view.end();
        }));
    }
  }

  SECTION("Keys are all found after inserting many keys.")
  {
    // Bulk insert keys
    thrust::for_each(thrust::device,
                     d_pairs.begin(),
                     d_pairs.end(),
                     [m_view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       m_view.insert(pair);
                     });

    SECTION("non-const view")
    {
      // All keys should be found
      REQUIRE(all_of(d_pairs.begin(),
                     d_pairs.end(),
                     [view] __device__(cuco::pair_type<Key, Value> const& pair) mutable {
                       auto const found = view.find(pair.first);
                       return (found != view.end()) and (found->first.load() == pair.first and
                                                         found->second.load() == pair.second);
                     }));
    }
    SECTION("const view")
    {
      // All keys should be found
      REQUIRE(all_of(
        d_pairs.begin(), d_pairs.end(), [view] __device__(cuco::pair_type<Key, Value> const& pair) {
          auto const found = view.find(pair.first);
          return (found != view.end()) and
                 (found->first.load() == pair.first and found->second.load() == pair.second);
        }));
    }
  }
}

template <typename MapType, int CAPACITY>
__global__ void shared_memory_test_kernel(
  typename MapType::device_view const* const device_views,
  typename MapType::device_view::key_type const* const insterted_keys,
  typename MapType::device_view::mapped_type const* const inserted_values,
  const size_t number_of_elements,
  bool* const keys_exist,
  bool* const keys_and_values_correct)
{
  // Each block processes one map
  const size_t map_id = blockIdx.x;
  const size_t offset = map_id * number_of_elements;

  __shared__ typename MapType::pair_atomic_type sm_buffer[CAPACITY];

  auto g = cg::this_thread_block();
  typename MapType::device_view sm_device_view =
    MapType::device_view::make_copy(g, sm_buffer, device_views[map_id]);

  for (int i = g.thread_rank(); i < number_of_elements; i += g.size()) {
    auto found_pair_it = sm_device_view.find(insterted_keys[offset + i]);

    if (found_pair_it != sm_device_view.end()) {
      keys_exist[offset + i] = true;
      if (found_pair_it->first == insterted_keys[offset + i] and
          found_pair_it->second == inserted_values[offset + i]) {
        keys_and_values_correct[offset + i] = true;
      } else {
        keys_and_values_correct[offset + i] = false;
      }
    } else {
      keys_exist[offset + i]              = false;
      keys_and_values_correct[offset + i] = true;
    }
  }
}

TEMPLATE_TEST_CASE_SIG("Shared memory static map",
                       "",
                       ((typename T, dist_type Dist), T, Dist),
                       (int32_t, dist_type::UNIQUE),
                       (int64_t, dist_type::UNIQUE),
                       (int32_t, dist_type::UNIFORM),
                       (int64_t, dist_type::UNIFORM),
                       (int32_t, dist_type::GAUSSIAN),
                       (int64_t, dist_type::GAUSSIAN))
{
  using KeyType                = T;
  using ValueType              = T;
  using MapType                = cuco::static_reduction_map<KeyType, ValueType>;
  using DeviceViewType         = typename MapType::device_view;
  using DeviceViewIteratorType = typename DeviceViewType::iterator;

  constexpr std::size_t number_of_maps  = 1000;
  constexpr std::size_t elements_in_map = 500;
  constexpr std::size_t map_capacity    = 2 * elements_in_map;

  // one array for all maps, first elements_in_map element belong to map 0, second to map 1 and so
  // on
  std::vector<KeyType> h_keys(number_of_maps * elements_in_map);
  std::vector<ValueType> h_values(number_of_maps * elements_in_map);
  std::vector<cuco::pair_type<KeyType, ValueType>> h_pairs(number_of_maps * elements_in_map);

  // using std::unique_ptr because static_reduction_map does not have copy/move
  // constructor/assignment operator yet
  std::vector<std::unique_ptr<MapType>> maps;

  for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
    const std::size_t offset = map_id * elements_in_map;

    generate_keys<Dist, KeyType>(h_keys.begin() + offset,
                                 h_keys.begin() + offset + elements_in_map);

    for (std::size_t i = 0; i < elements_in_map; ++i) {
      KeyType key                = h_keys[offset + i];
      ValueType val              = key < std::numeric_limits<KeyType>::max() ? key + 1 : 0;
      h_values[offset + i]       = val;
      h_pairs[offset + i].first  = key;
      h_pairs[offset + i].second = val;
    }

    maps.push_back(std::make_unique<MapType>(map_capacity, -1, -1));
  }

  thrust::device_vector<KeyType> d_keys(h_keys);
  thrust::device_vector<ValueType> d_values(h_values);
  thrust::device_vector<cuco::pair_type<KeyType, ValueType>> d_pairs(h_pairs);

  SECTION("Keys are all found after insertion.")
  {
    std::vector<DeviceViewType> h_device_views;
    for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
      const std::size_t offset = map_id * elements_in_map;

      MapType* map = maps[map_id].get();
      map->insert(d_pairs.begin() + offset, d_pairs.begin() + offset + elements_in_map);
      h_device_views.push_back(map->get_device_view());
    }
    thrust::device_vector<DeviceViewType> d_device_views(h_device_views);

    thrust::device_vector<bool> d_keys_exist(number_of_maps * elements_in_map);
    thrust::device_vector<bool> d_keys_and_values_correct(number_of_maps * elements_in_map);

    shared_memory_test_kernel<MapType, map_capacity>
      <<<number_of_maps, 64>>>(d_device_views.data().get(),
                               d_keys.data().get(),
                               d_values.data().get(),
                               elements_in_map,
                               d_keys_exist.data().get(),
                               d_keys_and_values_correct.data().get());

    REQUIRE(d_keys_exist.size() == d_keys_and_values_correct.size());
    auto zip = thrust::make_zip_iterator(
      thrust::make_tuple(d_keys_exist.begin(), d_keys_and_values_correct.begin()));

    REQUIRE(all_of(zip, zip + d_keys_exist.size(), [] __device__(auto const& z) {
      return thrust::get<0>(z) and thrust::get<1>(z);
    }));
  }

  SECTION("No key is found before insertion.")
  {
    std::vector<DeviceViewType> h_device_views;
    for (std::size_t map_id = 0; map_id < number_of_maps; ++map_id) {
      h_device_views.push_back(maps[map_id].get()->get_device_view());
    }
    thrust::device_vector<DeviceViewType> d_device_views(h_device_views);

    thrust::device_vector<bool> d_keys_exist(number_of_maps * elements_in_map);
    thrust::device_vector<bool> d_keys_and_values_correct(number_of_maps * elements_in_map);

    shared_memory_test_kernel<MapType, map_capacity>
      <<<number_of_maps, 64>>>(d_device_views.data().get(),
                               d_keys.data().get(),
                               d_values.data().get(),
                               elements_in_map,
                               d_keys_exist.data().get(),
                               d_keys_and_values_correct.data().get());

    REQUIRE(none_of(d_keys_exist.begin(), d_keys_exist.end(), [] __device__(const bool key_found) {
      return key_found;
    }));
  }
}