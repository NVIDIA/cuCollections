/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <test_utils.hpp>

#include <cuco/static_multimap.cuh>

#include <thrust/device_vector.h>
#include <thrust/distance.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

#include <catch2/catch_template_test_macros.hpp>

template <typename Map>
void test_multiplicity_two(Map& map, std::size_t num_items)
{
  using Key   = typename Map::key_type;
  using Value = typename Map::mapped_type;

  thrust::device_vector<Key> d_keys(num_items / 2);
  thrust::device_vector<cuco::pair<Key, Value>> d_pairs(num_items);

  thrust::sequence(thrust::device, d_keys.begin(), d_keys.end());
  // multiplicity = 2
  thrust::transform(thrust::device,
                    thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(num_items),
                    d_pairs.begin(),
                    [] __device__(auto i) {
                      return cuco::pair<Key, Value>{i / 2, i};
                    });

  thrust::device_vector<cuco::pair<Key, Value>> d_results(num_items);

  auto key_begin    = d_keys.begin();
  auto pair_begin   = d_pairs.begin();
  auto result_begin = d_results.begin();
  auto num_keys     = num_items / 2;
  thrust::device_vector<bool> d_contained(num_keys);

  SECTION("Non-inserted key/value pairs should not be contained.")
  {
    auto size = map.get_size();
    REQUIRE(size == 0);

    map.contains(key_begin, key_begin + num_keys, d_contained.begin());
    REQUIRE(cuco::test::none_of(d_contained.begin(), d_contained.end(), thrust::identity{}));
  }

  map.insert(pair_begin, pair_begin + num_items);

  SECTION("All inserted key/value pairs should be contained.")
  {
    auto size = map.get_size();
    REQUIRE(size == num_items);

    map.contains(key_begin, key_begin + num_keys, d_contained.begin());

    REQUIRE(cuco::test::all_of(d_contained.begin(), d_contained.end(), thrust::identity{}));
  }

  SECTION("Total count should be equal to the number of inserted pairs.")
  {
    // Count matching keys
    auto num = map.count(key_begin, key_begin + num_keys);

    REQUIRE(num == num_items);

    auto output_begin      = result_begin;
    auto output_end        = map.retrieve(key_begin, key_begin + num_keys, output_begin);
    std::size_t const size = thrust::distance(output_begin, output_end);

    REQUIRE(size == num_items);

    // sort before compare
    thrust::sort(
      thrust::device,
      d_results.begin(),
      d_results.end(),
      [] __device__(const cuco::pair<Key, Value>& lhs, const cuco::pair<Key, Value>& rhs) {
        if (lhs.first != rhs.first) { return lhs.first < rhs.first; }
        return lhs.second < rhs.second;
      });

    REQUIRE(
      cuco::test::equal(pair_begin,
                        pair_begin + num_items,
                        output_begin,
                        [] __device__(cuco::pair<Key, Value> lhs, cuco::pair<Key, Value> rhs) {
                          return lhs.first == rhs.first and lhs.second == rhs.second;
                        }));
  }

  SECTION("count and count_outer should return the same value.")
  {
    auto num       = map.count(key_begin, key_begin + num_keys);
    auto num_outer = map.count_outer(key_begin, key_begin + num_keys);

    REQUIRE(num == num_outer);
  }

  SECTION("Output of retrieve and retrieve_outer should be the same.")
  {
    auto output_begin = result_begin;
    auto output_end   = map.retrieve(key_begin, key_begin + num_keys, output_begin);
    auto size         = thrust::distance(output_begin, output_end);

    output_end      = map.retrieve_outer(key_begin, key_begin + num_keys, output_begin);
    auto size_outer = thrust::distance(output_begin, output_end);

    REQUIRE(size == size_outer);

    // sort before compare
    thrust::sort(
      thrust::device,
      d_results.begin(),
      d_results.end(),
      [] __device__(const cuco::pair<Key, Value>& lhs, const cuco::pair<Key, Value>& rhs) {
        if (lhs.first != rhs.first) { return lhs.first < rhs.first; }
        return lhs.second < rhs.second;
      });

    REQUIRE(
      cuco::test::equal(pair_begin,
                        pair_begin + num_items,
                        output_begin,
                        [] __device__(cuco::pair<Key, Value> lhs, cuco::pair<Key, Value> rhs) {
                          return lhs.first == rhs.first and lhs.second == rhs.second;
                        }));
  }
}

TEMPLATE_TEST_CASE_SIG(
  "Multiplicity equals two",
  "",
  ((typename Key, typename Value, cuco::test::probe_sequence Probe), Key, Value, Probe),
  (int32_t, int32_t, cuco::test::probe_sequence::linear_probing),
  (int32_t, int64_t, cuco::test::probe_sequence::linear_probing),
  (int64_t, int64_t, cuco::test::probe_sequence::linear_probing),
  (int32_t, int32_t, cuco::test::probe_sequence::double_hashing),
  (int32_t, int64_t, cuco::test::probe_sequence::double_hashing),
  (int64_t, int64_t, cuco::test::probe_sequence::double_hashing))
{
  constexpr std::size_t num_items{4};

  using probe =
    std::conditional_t<Probe == cuco::test::probe_sequence::linear_probing,
                       cuco::legacy::linear_probing<1, cuco::default_hash_function<Key>>,
                       cuco::legacy::double_hashing<8, cuco::default_hash_function<Key>>>;

  cuco::static_multimap<Key, Value, cuda::thread_scope_device, cuco::cuda_allocator<char>, probe>
    map{5, cuco::empty_key<Key>{-1}, cuco::empty_value<Value>{-1}};
  test_multiplicity_two(map, num_items);
}
