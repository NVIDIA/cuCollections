/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utils.hpp>

#include <cuco/static_multimap.cuh>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <catch2/catch_template_test_macros.hpp>

// Custom pair equal
template <typename Key, typename Value>
struct pair_equal {
  __device__ bool operator()(const cuco::pair_type<Key, Value>& lhs,
                             const cuco::pair_type<Key, Value>& rhs) const
  {
    return lhs.first == rhs.first;
  }
};

template <typename Key, typename Value, typename Map, typename PairIt>
__inline__ void test_pair_functions(Map& map, PairIt pair_begin, std::size_t num_pairs)
{
  map.insert(pair_begin, pair_begin + num_pairs);
  CUCO_CUDA_TRY(hipStreamSynchronize(0));

  auto res = map.get_size();
  REQUIRE(res == num_pairs);

  // query pair matching rate = 50%
  thrust::transform(thrust::device,
                    thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(num_pairs),
                    pair_begin,
                    [] __device__(auto i) {
                      return cuco::pair_type<Key, Value>{i, i};
                    });

  SECTION("pair_contains returns true for all inserted pairs and false for non-inserted ones.")
  {
    thrust::device_vector<bool> result(num_pairs);
    auto res_begin = result.begin();
    map.pair_contains(pair_begin, pair_begin + num_pairs, res_begin, pair_equal<Key, Value>{});

    auto true_iter  = thrust::make_constant_iterator(true);
    auto false_iter = thrust::make_constant_iterator(false);

    REQUIRE(
      cuco::test::equal(res_begin, res_begin + num_pairs / 2, true_iter, thrust::equal_to<bool>{}));
    REQUIRE(cuco::test::equal(
      res_begin + num_pairs / 2, res_begin + num_pairs, false_iter, thrust::equal_to<bool>{}));
  }

  SECTION("Output of pair_count and pair_retrieve should be coherent.")
  {
    auto num = map.pair_count(pair_begin, pair_begin + num_pairs, pair_equal<Key, Value>{});

    auto out1_begin = thrust::make_zip_iterator(
      thrust::make_tuple(thrust::make_discard_iterator(), thrust::make_discard_iterator()));
    auto out2_begin = thrust::make_zip_iterator(
      thrust::make_tuple(thrust::make_discard_iterator(), thrust::make_discard_iterator()));

    REQUIRE(num == num_pairs);

    auto [out1_end, out2_end] = map.pair_retrieve(
      pair_begin, pair_begin + num_pairs, out1_begin, out2_begin, pair_equal<Key, Value>{});
    std::size_t const size = std::distance(out2_begin, out1_end);

    REQUIRE(size == num_pairs);
  }

  SECTION("Output of pair_count_outer and pair_retrieve_outer should be coherent.")
  {
    auto num = map.pair_count_outer(pair_begin, pair_begin + num_pairs, pair_equal<Key, Value>{});

    auto out1_begin = thrust::make_zip_iterator(
      thrust::make_tuple(thrust::make_discard_iterator(), thrust::make_discard_iterator()));
    auto out2_begin = thrust::make_zip_iterator(
      thrust::make_tuple(thrust::make_discard_iterator(), thrust::make_discard_iterator()));

    REQUIRE(num == (num_pairs + num_pairs / 2));

    auto [out1_end, out2_end] = map.pair_retrieve_outer(
      pair_begin, pair_begin + num_pairs, out1_begin, out2_begin, pair_equal<Key, Value>{});
    std::size_t const size = std::distance(out1_begin, out1_end);

    REQUIRE(size == (num_pairs + num_pairs / 2));
  }
}

TEMPLATE_TEST_CASE_SIG(
  "Tests of pair functions",
  "",
  ((typename Key, typename Value, cuco::test::probe_sequence Probe), Key, Value, Probe),
  (int32_t, int32_t, cuco::test::probe_sequence::linear_probing),
  (int32_t, int64_t, cuco::test::probe_sequence::linear_probing),
  (int64_t, int64_t, cuco::test::probe_sequence::linear_probing),
  (int32_t, int32_t, cuco::test::probe_sequence::double_hashing),
  (int32_t, int64_t, cuco::test::probe_sequence::double_hashing),
  (int64_t, int64_t, cuco::test::probe_sequence::double_hashing))
{
  constexpr std::size_t num_pairs{4};
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(num_pairs);

  // pair multiplicity = 2
  thrust::transform(thrust::device,
                    thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(num_pairs),
                    d_pairs.begin(),
                    [] __device__(auto i) {
                      return cuco::pair_type<Key, Value>{i / 2, i};
                    });

  using probe = std::conditional_t<
    Probe == cuco::test::probe_sequence::linear_probing,
    cuco::linear_probing<1, cuco::murmurhash3_32<Key>>,
    cuco::double_hashing<8, cuco::murmurhash3_32<Key>, cuco::murmurhash3_32<Key>>>;

  cuco::static_multimap<Key, Value, cuda::thread_scope_device, cuco::cuda_allocator<char>, probe>
    map{num_pairs * 2, cuco::empty_key<Key>{-1}, cuco::empty_value<Value>{-1}};
  test_pair_functions<Key, Value>(map, d_pairs.begin(), num_pairs);
}
