/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <test_utils.hpp>

#include <cuco/static_multimap.cuh>

#include <cuda/functional>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/sequence.h>

#include <catch2/catch_template_test_macros.hpp>

template <typename Map>
void test_insert(Map& map, std::size_t num_keys)
{
  using Key   = typename Map::key_type;
  using Value = typename Map::mapped_type;

  thrust::device_vector<Key> d_keys(num_keys);

  thrust::sequence(thrust::device, d_keys.begin(), d_keys.end());

  auto keys_begin  = d_keys.begin();
  auto pairs_begin = thrust::make_transform_iterator(
    thrust::make_counting_iterator(0),
    cuda::proclaim_return_type<cuco::pair<Key, Value>>(
      [] __device__(auto i) { return cuco::pair<Key, Value>{i, i}; }));
  thrust::device_vector<bool> d_contained(num_keys);

  SECTION("Non-inserted keys should not be contained.")
  {
    map.contains(keys_begin, keys_begin + num_keys, d_contained.begin());
    REQUIRE(cuco::test::none_of(d_contained.begin(), d_contained.end(), thrust::identity{}));
  }

  map.insert(pairs_begin, pairs_begin + num_keys);

  SECTION("All inserted keys should be contained.")
  {
    map.contains(keys_begin, keys_begin + num_keys, d_contained.begin());
    REQUIRE(cuco::test::all_of(d_contained.begin(), d_contained.end(), thrust::identity{}));
  }
}

TEMPLATE_TEST_CASE_SIG(
  "static_multimap insert test",
  "",
  ((typename Key, typename Value, cuco::test::probe_sequence Probe, int CGSize),
   Key,
   Value,
   Probe,
   CGSize),
  (int32_t, int32_t, cuco::test::probe_sequence::double_hashing, 1),
  (int32_t, int64_t, cuco::test::probe_sequence::double_hashing, 1),
  (int32_t, int32_t, cuco::test::probe_sequence::double_hashing, 2),
  (int32_t, int64_t, cuco::test::probe_sequence::double_hashing, 2),
  (int64_t, int32_t, cuco::test::probe_sequence::double_hashing, 1),
  (int64_t, int64_t, cuco::test::probe_sequence::double_hashing, 1),
  (int64_t, int32_t, cuco::test::probe_sequence::double_hashing, 2),
  (int64_t, int64_t, cuco::test::probe_sequence::double_hashing, 2),
  (int32_t, int32_t, cuco::test::probe_sequence::linear_probing, 1),
  (int32_t, int64_t, cuco::test::probe_sequence::linear_probing, 1),
  (int32_t, int32_t, cuco::test::probe_sequence::linear_probing, 2),
  (int32_t, int64_t, cuco::test::probe_sequence::linear_probing, 2),
  (int64_t, int32_t, cuco::test::probe_sequence::linear_probing, 1),
  (int64_t, int64_t, cuco::test::probe_sequence::linear_probing, 1),
  (int64_t, int32_t, cuco::test::probe_sequence::linear_probing, 2),
  (int64_t, int64_t, cuco::test::probe_sequence::linear_probing, 2))
{
  constexpr std::size_t num_keys{4'000};

  using extent_type = cuco::extent<std::size_t>;
  using probe       = std::conditional_t<
          Probe == cuco::test::probe_sequence::linear_probing,
          cuco::linear_probing<CGSize, cuco::murmurhash3_32<Key>>,
          cuco::double_hashing<CGSize, cuco::murmurhash3_32<Key>, cuco::murmurhash3_32<Key>>>;

  auto map = cuco::experimental::static_multimap<Key,
                                                 Value,
                                                 extent_type,
                                                 cuda::thread_scope_device,
                                                 thrust::equal_to<Key>,
                                                 probe,
                                                 cuco::cuda_allocator<std::byte>,
                                                 cuco::storage<2>>{
    extent_type{num_keys}, cuco::empty_key<Key>{-1}, cuco::empty_value<Value>{-1}};

  test_insert(map, num_keys);
}
