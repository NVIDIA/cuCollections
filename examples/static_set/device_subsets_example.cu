#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuco/static_set_ref.cuh>
#include <cuco/storage.cuh>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>

#include <hip/hip_cooperative_groups.h>

#include <cuda/std/array>

#include <algorithm>
#include <cstddef>
#include <iostream>

/**
 * @file device_subsets_example.cu
 * @brief Demonstrates how to use one bulk set storage to create multiple subsets and perform
 * individual operations via device-side ref APIs.
 *
 * To optimize memory usage, especially when dealing with expensive data allocation and multiple
 * hashsets, a practical solution involves employing a single bulk storage for generating subsets.
 * This eliminates the need for separate memory allocation and deallocation for each container. This
 * can be achieved by using the lightweight non-owning ref type.
 *
 * @note This example is for demonstration purposes only. It is not intended to show the most
 * performant way to do the example algorithm.
 */

auto constexpr cg_size     = 8;   ///< A CUDA Cooperative Group of 8 threads to handle each subset
auto constexpr window_size = 1;   ///< Number of concurrent slots handled by each thread
auto constexpr N           = 10;  ///< Number of elements to insert and query

using key_type            = int;  ///< Key type
using probing_scheme_type = cuco::experimental::linear_probing<
  cg_size,
  cuco::default_hash_function<key_type>>;  ///< Type controls CG granularity and probing scheme
                                           ///< (linear probing v.s. double hashing)
using storage_type = cuco::experimental::aow_storage<key_type, window_size>;  ///< Storage type
using storage_ref_type =
  typename storage_type::ref_type;  ///< Lightweight non-owning storage ref type
template <typename Operator>
using ref_type = cuco::experimental::static_set_ref<key_type,
                                                    cuda::thread_scope_device,
                                                    thrust::equal_to<key_type>,
                                                    probing_scheme_type,
                                                    storage_ref_type,
                                                    Operator>;  ///< Set ref type

/// Sample data to insert and query
__device__ constexpr std::array<key_type, N> data = {1, 3, 5, 7, 9, 11, 13, 15, 17, 19};
/// Empty slots are represented by reserved "sentinel" values. These values should be selected such
/// that they never occur in your input data.
key_type constexpr empty_key_sentinel = -1;

/**
 * @brief Inserts sample data into subsets by using cooperative group
 *
 * Each Cooperative Group creates its own subset and inserts `N` sample data.
 *
 * @tparam WindowType Storage window type
 * @tparam SizeType Size type
 * @tparam OffsetType Offset type
 *
 * @param windows Pointer to the window array
 * @param sizes Pointer to the subset sizes array
 * @param offsets Pointer to the subset offsets array
 */
template <typename WindowType, typename SizeType, typename OffsetType>
__global__ void insert(WindowType* windows, SizeType* sizes, OffsetType* offsets)
{
  namespace cg = cooperative_groups;

  auto const tile = cg::tiled_partition<cg_size>(cg::this_thread_block());
  // Get subset (or CG) index
  auto const idx = (blockDim.x * blockIdx.x + threadIdx.x) / cg_size;

  // Construct an "insert" ref with the given storage
  auto set_ref = ref_type<cuco::experimental::insert_tag>{
    cuco::empty_key<key_type>{-1}, {}, {}, storage_ref_type{sizes[idx], windows + offsets[idx]}};

  // Insert `N` elemtns into the set with CG insert
  for (int i = 0; i < N; i++) {
    set_ref.insert(tile, data[i]);
  }
}

/**
 * @brief All inserted data can be found
 *
 * Each Cooperative Group reconstructs its own subset ref based on the storage parameters and
 * verifies all inserted data can be found.
 *
 * @tparam WindowType Storage window type
 * @tparam SizeType Size type
 * @tparam OffsetType Offset type
 *
 * @param windows Pointer to the window array
 * @param sizes Pointer to the subset sizes array
 * @param offsets Pointer to the subset offsets array
 */
template <typename WindowType, typename SizeType, typename OffsetType>
__global__ void find(WindowType* windows, SizeType* sizes, OffsetType* offsets)
{
  namespace cg = cooperative_groups;

  auto const tile = cg::tiled_partition<cg_size>(cg::this_thread_block());
  auto const idx  = (blockDim.x * blockIdx.x + threadIdx.x) / cg_size;

  // Reconstruct an "find" ref with the same storage
  auto set_ref = ref_type<cuco::experimental::find_tag>{
    cuco::empty_key<key_type>{-1}, {}, {}, storage_ref_type{sizes[idx], windows + offsets[idx]}};

  // Result denoting if any of the inserted data is not found
  __shared__ int result;
  if (threadIdx.x == 0) { result = 0; }
  __syncthreads();

  for (int i = 0; i < N; i++) {
    // Query the set with inserted data
    auto const found = set_ref.find(tile, data[i]);
    // Record if the inserted data has been found
    atomicOr(&result, *found != data[i]);
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    // If the result is still 0, all inserted data are found.
    if (result == 0) { printf("Success! Found all inserted elements.\n"); }
  }
}

int main()
{
  // Number of subsets to be created
  auto constexpr num = 16;
  // Each subset may have a different requested size
  auto constexpr subset_sizes =
    std::array<std::size_t, num>{20, 20, 20, 20, 30, 30, 30, 30, 40, 40, 40, 40, 50, 50, 50, 50};

  auto valid_sizes = std::vector<std::size_t>(num);
  // Compute the valid sizes based on requested sizes
  std::generate(valid_sizes.begin(), valid_sizes.end(), [&, n = 0]() mutable {
    // The requested size could cause infinite probing sequences for hash sets thus the valid size
    // required by the container MUST be computed via `make_window_extent`
    return cuco::experimental::make_window_extent<cg_size, window_size>(subset_sizes[n++]);
  });

  // Copy host data to device
  auto const d_sizes = thrust::device_vector<std::size_t>{valid_sizes};
  auto d_offsets     = thrust::device_vector<std::size_t>(num);
  // Compute the offset for each subset
  thrust::exclusive_scan(d_sizes.begin(), d_sizes.end(), d_offsets.begin());

  // Get the total size of all subsets.
  auto const num_windows = thrust::reduce(valid_sizes.begin(), valid_sizes.end());

  // Create a single bulk storage used by all subsets
  auto d_set_storage = storage_type{num_windows};
  // Initializes the storage with the given sentinel
  d_set_storage.initialize(empty_key_sentinel);

  // Insert sample data
  insert<<<1, 128>>>(d_set_storage.data(), d_sizes.data().get(), d_offsets.data().get());
  // Find all inserted data
  find<<<1, 128>>>(d_set_storage.data(), d_sizes.data().get(), d_offsets.data().get());

  return 0;
}
