#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuco/static_set_ref.cuh>
#include <cuco/storage.cuh>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>

#include <hip/hip_cooperative_groups.h>

#include <cuda/std/array>

#include <algorithm>
#include <cstddef>
#include <iostream>

auto constexpr cg_size     = 8;   ///< A CUDA Cooperative Group of 8 threads to handle each subset
auto constexpr window_size = 1;   ///< TODO: how to explain window size (vector length) to users
auto constexpr N           = 10;  ///< Number of elements to insert and query

using key_type = int;
using probing_scheme_type =
  cuco::experimental::linear_probing<cg_size, cuco::default_hash_function<key_type>>;
using storage_ref_type = cuco::experimental::aow_storage_ref<key_type, window_size>;
template <typename Operator>
using ref_type = cuco::experimental::static_set_ref<key_type,
                                                    cuda::thread_scope_device,
                                                    thrust::equal_to<key_type>,
                                                    probing_scheme_type,
                                                    storage_ref_type,
                                                    Operator>;

/// data to insert/query
__device__ constexpr std::array<key_type, N> data = {1, 3, 5, 7, 9, 11, 13, 15, 17, 19};
/// Empty slots are represented by reserved "sentinel" values. These values should be selected such
/// that they never occur in your input data.
key_type constexpr empty_key_sentinel = -1;

template <typename WindowT>
__global__ void initialize(WindowT* windows, std::size_t n, typename WindowT::value_type value)
{
  using T = typename WindowT::value_type;

  auto const loop_stride = gridDim.x * blockDim.x;
  auto idx               = blockDim.x * blockIdx.x + threadIdx.x;

  while (idx < n) {
    auto& window_slots = *(windows + idx);
#pragma unroll
    for (auto& slot : window_slots) {
      new (&slot) T{value};
    }
    idx += loop_stride;
  }
}

// insert a set of keys into a hash set using one cooperative group for each task
template <typename Window, typename Size, typename Offset>
__global__ void insert(Window* windows, Size* sizes, Offset* offsets)
{
  namespace cg = cooperative_groups;

  auto const tile = cg::tiled_partition<cg_size>(cg::this_thread_block());
  auto const idx  = (blockDim.x * blockIdx.x + threadIdx.x) / cg_size;

  auto set_ref = ref_type<cuco::experimental::insert_tag>{
    cuco::empty_key<key_type>{-1}, {}, {}, storage_ref_type{sizes[idx], windows + offsets[idx]}};

  // Each cooperative_groups inserts all elements in `data` into its own subset
  for (int i = 0; i < N; i++) {
    set_ref.insert(tile, data[i]);
  }
}

// insert a set of keys into a hash set using one cooperative group for each task
template <typename Window, typename Size, typename Offset>
__global__ void find(Window* windows, Size* sizes, Offset* offsets)
{
  namespace cg = cooperative_groups;

  auto const tile = cg::tiled_partition<cg_size>(cg::this_thread_block());
  auto const idx  = (blockDim.x * blockIdx.x + threadIdx.x) / cg_size;

  auto set_ref = ref_type<cuco::experimental::find_tag>{
    cuco::empty_key<key_type>{-1}, {}, {}, storage_ref_type{sizes[idx], windows + offsets[idx]}};

  __shared__ int result;
  if (threadIdx.x == 0) { result = 0; }
  __syncthreads();

  for (int i = 0; i < N; i++) {
    auto const found = set_ref.find(tile, data[i]);
    // Record if the inserted data has been found
    atomicOr(&result, *found != data[i]);
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    if (result == 0) { printf("Success! Found all inserted elements.\n"); }
  }
}

/**
 * @file device_subsets_example.cu
 * @brief Demonstrates usage of the static_set device-side APIs.
 *
 * static_set provides a non-owning reference which can be used to interact with
 * the container from within device code.
 */
int main()
{
  // Number of subsets
  auto constexpr num = 16;
  // Sizes of the 16 subsets to be created on the device
  auto constexpr subset_sizes =
    std::array<std::size_t, num>{20, 20, 20, 20, 30, 30, 30, 30, 40, 40, 40, 40, 50, 50, 50, 50};

  auto valid_sizes = std::vector<std::size_t>(num);
  std::generate(valid_sizes.begin(), valid_sizes.end(), [&, n = 0]() mutable {
    return cuco::experimental::make_window_extent<cg_size, window_size>(subset_sizes[n++]);
  });

  auto const d_sizes = thrust::device_vector<std::size_t>{valid_sizes};
  auto d_offsets     = thrust::device_vector<std::size_t>(num);
  thrust::exclusive_scan(d_sizes.begin(), d_sizes.end(), d_offsets.begin());

  auto const num_windows = thrust::reduce(valid_sizes.begin(), valid_sizes.end());

  // One allocation for all subsets
  auto d_set_storage = cuco::experimental::aow_storage<key_type, window_size>{num_windows};
  // Initializes the storage with the given sentinel
  d_set_storage.initialize(empty_key_sentinel);

  insert<<<1, 128>>>(d_set_storage.data(), d_sizes.data().get(), d_offsets.data().get());
  find<<<1, 128>>>(d_set_storage.data(), d_sizes.data().get(), d_offsets.data().get());

  return 0;
}
