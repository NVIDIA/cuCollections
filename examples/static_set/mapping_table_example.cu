/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuco/static_set.cuh>

#include <cuda/std/array>
#include <cuda/std/tuple>
#include <thrust/device_vector.h>

#include <iostream>

/**
 * @file mapping_table_example.cu
 *
 * @brief Demonstrates how to use hash set as a lookup table of the original data
 *
 * `cuco` hash tables such as `cuco::static_set` or `cuco::static_map` currently support only 4/8
 * byte keys. This limitation arises because `cuco` hash tables rely on atomic Compare-And-Swap
 * (CAS) operations for key insertions (or queries), and the hardware natively supports only 4-byte
 * and 8-byte CAS. To enable support for larger keys, one approach is to implement atomic lock
 * tables at the software level. However, this approach would lead to a notable performance decrease
 * due to the high runtime cost of atomic lock tables.
 *
 * Additionally, `cuco` hash tables use open addressing as the hash collision resolution method.
 * This approach requires users to provide a sentinel that indicates unused slots in the data
 * structure. The sentinel value is a reserved value that must be never present in the problem. Note
 * that inserting or querying a sentinel value is undefined behavior. This can be problematic,
 * especially when the input data type is complex and determining a valid sentinel value is not
 * straightforward.
 *
 * This sample code demonstrates a solution to address these issues by using hash set as an
 * indirection mapping table to the original data:
 *  - The keys inserted in the hash table are indices of the original data array.
 *  - Using `-1` as a sentinel value is safe because accessing `data[-1]` is invalid.
 *  - Custom hashers and key equality comparators are required to hash and compare original keys
 *    based on indices.
 */

/**
 * @brief User-defined key equal to compare two keys
 *
 * @tparam T Original key type
 */
template <typename T>
struct my_equal {
  my_equal(T const* data) : _data{data} {}
  /**
   * @brief Checks if two keys are identical based on their indices in the
   * original data array
   *
   * @param lhs The left hand side index
   * @param rhs The right hand side index
   * @return 'true' if two tuples are indentical
   */
  __device__ constexpr bool operator()(int32_t lhs, int32_t rhs) const
  {
    // Check all 4 elements of a tuple to determine if two tuples are identical
    return cuda::std::get<0>(_data[lhs]) == cuda::std::get<0>(_data[rhs]) and
           cuda::std::get<1>(_data[lhs]) == cuda::std::get<1>(_data[rhs]) and
           cuda::std::get<2>(_data[lhs]) == cuda::std::get<2>(_data[rhs]) and
           cuda::std::get<3>(_data[lhs]) == cuda::std::get<3>(_data[rhs]);
  }
  T const* _data;
};

/**
 * @brief User-defined hash function to hash the original data based on its index
 *
 * @tparam T Original key type
 */
template <typename T>
struct my_hasher {
  my_hasher(T const* data) : _data{data} {}
  __device__ auto operator()(int32_t index) const
  {
    // Only hashes the first element of a tuple for demonstrataion purposes
    return cuda::std::get<0>(_data[index]);
  }
  T const* _data;
};

/**
 * @brief Utility to print the content of a given `tuple`
 *
 * @tparam T Type of the tuple
 */
template <typename T>
void print(T const& tuple)
{
  std::cout << "[" << cuda::std::get<0>(tuple) << ", " << cuda::std::get<1>(tuple) << ", "
            << cuda::std::get<2>(tuple) << ", "
            << "[" << cuda::std::get<3>(tuple)[0] << ", " << cuda::std::get<3>(tuple)[1] << ", "
            << cuda::std::get<3>(tuple)[2] << ", " << cuda::std::get<3>(tuple)[3] << "]]\n";
}

int main(void)
{
  // The original key type is larger than 8-byte and complex to spell the full type name. In the
  // meanwhile, it's not obvious to determine a valid sentinel value without instrospecting the data
  using Key = cuda::std::tuple<uint32_t, char, bool, cuda::std::array<double, 4UL>>;
  auto const h_data =
    std::vector<Key>{cuda::std::tuple{11u, 'a', true, cuda::std::array{1., 2., 3., 4.}},
                     cuda::std::tuple{11u, 'a', true, cuda::std::array{1., 2., 3., 4.}},
                     cuda::std::tuple{22u, 'b', true, cuda::std::array{5., 6., 7., 8.}},
                     cuda::std::tuple{11u, 'a', true, cuda::std::array{5., 6., 7., 8.}},
                     cuda::std::tuple{11u, 'a', false, cuda::std::array{1., 2., 3., 4.}}};
  auto const size = h_data.size();
  thrust::device_vector<Key> d_data{h_data};

  // The actual key type is an index type, `int32_t` is large enough to cover the whole input range
  // and 4-byte atomic CAS is more efficient than the 8-byte one.
  using ActualKey = int32_t;
  // `-1` is a valid sentinel value since one will never access `data[-1]`
  ActualKey constexpr empty_key_sentinel = -1;

  auto const data_ptr = d_data.data().get();
  auto set = cuco::static_set{cuco::extent<std::size_t>{size * 2},  // about 50% load factor
                              cuco::empty_key{empty_key_sentinel},
                              my_equal{data_ptr},
                              cuco::linear_probing<1, my_hasher<Key>>{my_hasher<Key>{data_ptr}}};

  // The actual keys are indices of 5 elements
  auto const actual_keys = thrust::device_vector<ActualKey>{0, 1, 2, 3, 4};
  set.insert(actual_keys.begin(), actual_keys.end());

  auto unique_keys           = thrust::device_vector<ActualKey>(size);
  auto const unique_keys_end = set.retrieve_all(unique_keys.begin());
  auto const num             = std::distance(unique_keys.begin(), unique_keys_end);

  std::cout << "There are " << num << " distinct input elements:\n";
  for (auto i = 0; i < num; ++i) {
    // Retrieve query output based on indices
    print(h_data[unique_keys[i]]);
  }

  return 0;
}
