/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuco/static_map.cuh>

#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/logical.h>
#include <thrust/transform.h>

#include <cuda/functional>

// User-defined key type
#if !defined(CUCO_HAS_INDEPENDENT_THREADS)
struct custom_key_type {
  int32_t a;
  int32_t b;

  __host__ __device__ custom_key_type() {}
  __host__ __device__ custom_key_type(int32_t x) : a{x}, b{x} {}
};
#else
// Key type larger than 8B only supported for sm_70 and up
struct custom_key_type {
  int32_t a;
  int32_t b;
  int32_t c;

  __host__ __device__ custom_key_type() {}
  __host__ __device__ custom_key_type(int32_t x) : a{x}, b{x}, c{x} {}

  // Device equality operator is mandatory due to libcudacxx bug:
  // https://github.com/NVIDIA/libcudacxx/issues/223
  __device__ bool operator==(custom_key_type const& other) const
  {
    return a == other.a and b == other.b and c == other.c;
  }
};
#endif

// User-defined value type
struct custom_value_type {
  int32_t f;
  int32_t s;

  __host__ __device__ custom_value_type() {}
  __host__ __device__ custom_value_type(int32_t x) : f{x}, s{x} {}
};

// User-defined device hash callable
struct custom_hash {
  template <typename key_type>
  __device__ uint32_t operator()(key_type k)
  {
    return k.a;
  };
};

// User-defined device key equal callable
struct custom_key_equals {
  template <typename key_type>
  __device__ bool operator()(key_type const& lhs, key_type const& rhs)
  {
    return lhs.a == rhs.a;
  }
};

int main(void)
{
  constexpr std::size_t num_pairs = 80'000;

  // Set emtpy sentinels
  auto const empty_key_sentinel   = custom_key_type{-1};
  auto const empty_value_sentinel = custom_value_type{-1};

  // Create an iterator of input key/value pairs
  auto pairs_begin = thrust::make_transform_iterator(
    thrust::make_counting_iterator<int32_t>(0),
    cuda::proclaim_return_type<cuco::pair<custom_key_type, custom_value_type>>(
      [] __device__(auto i) { return cuco::make_pair(custom_key_type{i}, custom_value_type{i}); }));

  // Construct a map with 100,000 slots using the given empty key/value sentinels. Note the
  // capacity is chosen knowing we will insert 80,000 keys, for an load factor of 80%.
  cuco::static_map<custom_key_type, custom_value_type> map{
    100'000, cuco::empty_key{empty_key_sentinel}, cuco::empty_value{empty_value_sentinel}};

  // Inserts 80,000 pairs into the map by using the custom hasher and custom equality callable
  map.insert(pairs_begin, pairs_begin + num_pairs, custom_hash{}, custom_key_equals{});

  // Reproduce inserted keys
  auto insert_keys =
    thrust::make_transform_iterator(thrust::make_counting_iterator<int32_t>(0),
                                    cuda::proclaim_return_type<custom_key_type>(
                                      [] __device__(auto i) { return custom_key_type{i}; }));

  thrust::device_vector<bool> contained(num_pairs);

  // Determine if all the inserted keys can be found by using the same hasher and equality
  // function as `insert`. If a key `insert_keys[i]` doesn't exist, `contained[i] == false`.
  map.contains(
    insert_keys, insert_keys + num_pairs, contained.begin(), custom_hash{}, custom_key_equals{});
  // This will fail due to inconsistent hash and key equal.
  // map.contains(insert_keys, insert_keys + num_pairs, contained.begin());

  // All inserted keys are contained
  assert(
    thrust::all_of(contained.begin(),
                   contained.end(),
                   cuda::proclaim_return_type<bool>([] __device__(auto const& b) { return b; })));

  return 0;
}
