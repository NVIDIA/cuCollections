#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cuco/distinct_count_estimator.cuh>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#include <cmath>
#include <cstddef>
#include <iostream>

/**
 * @file host_bulk_example.cu
 * @brief Demonstrates usage of `cuco::distinct_count_estimator` "bulk" host APIs.
 */
int main(void)
{
  using T                         = int;
  constexpr std::size_t num_items = 1ull << 28;  // 1GB

  thrust::device_vector<T> items(num_items);

  // Generate `num_items` distinct items
  thrust::sequence(items.begin(), items.end(), 0);

  // We define the desired standard deviation of the approximation error
  // 0.0122197 is the default value and corresponds to a 32KB sketch size
  auto const sd = cuco::standard_deviation{0.0122197};

  // Initialize the estimator
  cuco::distinct_count_estimator<T> estimator{sd};

  // Add all items to the estimator
  estimator.add(items.begin(), items.end());

  // Adding the same items again will not affect the result
  estimator.add(items.begin(), items.begin() + num_items / 2);

  // Calculate the cardinality estimate
  std::size_t const estimated_cardinality = estimator.estimate();

  std::cout << "True cardinality: " << num_items
            << "\nEstimated cardinality: " << estimated_cardinality << "\nError: "
            << std::abs(
                 static_cast<double>(estimated_cardinality) / static_cast<double>(num_items) - 1.0)
            << std::endl;

  return 0;
}