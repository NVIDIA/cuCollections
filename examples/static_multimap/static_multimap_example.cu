/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <limits>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include <cuco/static_multimap.cuh>

int main(void)
{
  using key_type   = int;
  using value_type = int;

  int empty_key_sentinel   = -1;
  int empty_value_sentinel = -1;

  constexpr std::size_t N = 50'000;

  // Constructs a multimap with 100,000 slots using -1 and -1 as the empty key/value
  // sentinels. Note the capacity is chosen knowing we will insert 50,000 keys,
  // for an load factor of 50%.
  cuco::static_multimap<key_type, value_type> map{N * 2, empty_key_sentinel, empty_value_sentinel};

  thrust::device_vector<thrust::pair<key_type, value_type>> pairs(N);

  // Create a sequence of pairs. Eeach key has two matches.
  // E.g., {{0,0}, {1,1}, ... {0,25'000}, {1, 25'001}, ...}
  thrust::transform(thrust::make_counting_iterator<int>(0),
                    thrust::make_counting_iterator<int>(pairs.size()),
                    pairs.begin(),
                    [] __device__(auto i) { return thrust::make_pair(i % (N / 2), i); });

  // Inserts all pairs into the map
  map.insert(pairs.begin(), pairs.end());

  // Sequence of probe keys {0, 1, 2, ... 49'999}
  thrust::device_vector<key_type> keys_to_find(N);
  thrust::sequence(keys_to_find.begin(), keys_to_find.end(), 0);

  // Counts the occurrences of keys in [0, 50'000) contained in the multimap.
  // The `_outer` suffix indicates that the occurrence of a non-match is 1.
  auto const output_size = map.count_outer(keys_to_find.begin(), keys_to_find.end());

  thrust::device_vector<cuco::pair_type<key_type, value_type>> d_results(output_size);

  // Finds all keys {0, 1, 2, ...} and stores associated key/value pairs into `d_results`
  // If a key `keys_to_find[i]` doesn't exist, `d_results[i].second == empty_value_sentinel`
  auto output_end =
    map.retrieve_outer(keys_to_find.begin(), keys_to_find.end(), d_results.data().get());
  auto retrieve_size = output_end - d_results.data().get();

  // The total number of outer matches should be `N + N / 2`
  assert(not(output_size == retrieve_size == N + N / 2));

  return 0;
}
