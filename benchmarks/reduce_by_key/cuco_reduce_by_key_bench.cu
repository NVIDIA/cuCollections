#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuco/detail/utils.hpp>
#include <cuco/static_reduction_map.cuh>
#include <key_generator.hpp>
#include <nvbench/nvbench.cuh>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <utils.hpp>

/**
 * @brief Enum representation for reduction operators
 */
enum class op_type { REDUCE_ADD, CUSTOM_OP };

NVBENCH_DECLARE_ENUM_TYPE_STRINGS(
  // Enum type:
  op_type,
  // Callable to generate input strings:
  // Short identifier used for tables, command-line args, etc.
  // Used when context is available to figure out the enum type.
  [](op_type o) {
    switch (o) {
      case op_type::REDUCE_ADD: return "REDUCE_ADD";
      case op_type::CUSTOM_OP: return "CUSTOM_OP";
      default: return "ERROR";
    }
  },
  // Callable to generate descriptions:
  // If non-empty, these are used in `--list` to describe values.
  // Used when context may not be available to figure out the type from the
  // input string.
  // Just use `[](auto) { return std::string{}; }` if you don't want these.
  [](auto) { return std::string{}; })

/**
 * @brief Maps the enum value of a cuco reduction operator to its actual type
 */
template <op_type Op>
struct op_type_map {
};

template <>
struct op_type_map<op_type::REDUCE_ADD> {
  template <typename T>
  using type = cuco::reduce_add<T>;
};

template <>
struct op_type_map<op_type::CUSTOM_OP> {
  template <typename T>
  using type = cuco::custom_op<T, 0, thrust::plus<T>>;  // sum reduction with CAS loop
};

/**
 * @brief A benchmark evaluating CUCO's reduce-by-key performance.
 */
template <typename Key, typename Value, op_type Op>
void nvbench_cuco_static_reduction_map_reduce_by_key(
  nvbench::state& state, nvbench::type_list<Key, Value, nvbench::enum_type<Op>>)
{
  using map_type = cuco::static_reduction_map<typename op_type_map<Op>::type<Value>, Key, Value>;

  auto const num_elems    = state.get_int64("NumInputs");
  auto const occupancy    = state.get_float64("Occupancy");
  auto const dist         = state.get_string("Distribution");
  auto const multiplicity = state.get_int64_or_default("Multiplicity", 8);

  std::vector<Key> h_keys(num_elems);
  std::vector<Value> h_values(num_elems);

  if (not generate_keys<Key>(dist, h_keys.begin(), h_keys.end(), multiplicity)) {
    state.skip("Invalid input distribution.");
    return;
  }

  // generate uniform random values
  generate_keys<Value>("UNIFORM", h_values.begin(), h_values.end(), 1);

  // the size of the hash table under a given target occupancy depends on the
  // number of unique keys in the input
  std::size_t const unique   = count_unique(h_keys.begin(), h_keys.end());
  std::size_t const capacity = std::ceil(SDIV(unique, occupancy));

  // alternative occupancy calculation based on the total number of inputs
  // std::size_t const capacity = num_elems / occupancy;

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<Value> d_values(h_values);

  auto d_pairs_begin =
    thrust::make_zip_iterator(thrust::make_tuple(d_keys.begin(), d_values.begin()));
  auto d_pairs_end = d_pairs_begin + num_elems;

  state.add_element_count(num_elems);

  state.exec(nvbench::exec_tag::sync | nvbench::exec_tag::timer,
             [&](nvbench::launch& launch, auto& timer) {
               map_type map{capacity, -1};

               timer.start();
               map.insert(d_pairs_begin, d_pairs_end, launch.get_stream());
               map.retrieve_all(d_keys.begin(), d_values.begin(), launch.get_stream());
               timer.stop();
             });
}

// type parameter dimensions for benchmark
using key_type_range   = nvbench::type_list<nvbench::int32_t, nvbench::int64_t>;
using value_type_range = nvbench::type_list<nvbench::int32_t, nvbench::int64_t>;
using op_type_range    = nvbench::enum_type_list<op_type::REDUCE_ADD>;

NVBENCH_BENCH_TYPES(nvbench_cuco_static_reduction_map_reduce_by_key,
                    NVBENCH_TYPE_AXES(key_type_range, value_type_range, op_type_range))
  .set_name("cuco_static_reduction_map_reduce_by_key_occupancy")
  .set_type_axes_names({"Key", "Value", "ReductionOp"})
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs
  .add_float64_axis("Occupancy", nvbench::range(0.5, 0.9, 0.1))  // occupancy range
  .add_int64_axis("Multiplicity", {8})  // only applies to uniform distribution
  .add_string_axis("Distribution", {"GAUSSIAN", "UNIFORM", "UNIQUE", "SAME"});

NVBENCH_BENCH_TYPES(nvbench_cuco_static_reduction_map_reduce_by_key,
                    NVBENCH_TYPE_AXES(key_type_range, value_type_range, op_type_range))
  .set_name("cuco_static_reduction_map_reduce_by_key_multiplicity")
  .set_type_axes_names({"Key", "Value", "ReductionOp"})
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs
  .add_float64_axis("Occupancy", {0.5, 0.8})   // fixed occupancy
  .add_int64_axis("Multiplicity",
                  {1, 10, 100, 1'000, 10'000, 100'000, 1'000'000})  // key multiplicity range
  .add_string_axis("Distribution", {"UNIFORM"});