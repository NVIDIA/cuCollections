/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuco/static_multimap.cuh>

#include <nvbench/nvbench.cuh>

#include <thrust/device_vector.h>

/**
 * @brief Generates input keys by a given number of repetitions per key.
 *
 */
template <typename Key, typename OutputIt>
static void generate_multikeys(OutputIt output_begin,
                               OutputIt output_end,
                               size_t const multiplicity)
{
  auto num_keys = std::distance(output_begin, output_end);

  for (auto i = 0; i < num_keys; ++i) {
    output_begin[i] = (i % (num_keys / multiplicity)) + 1;
  }
}

/**
 * @brief A benchmark evaluating multi-value retrieval performance by varing number of repetitions
 * per key:
 * - 100'000'000 keys are inserted
 * - Map occupancy is fixed at 0.4
 * - Number of repetitions per key: 1, ... , 128, 256
 *
 */
template <typename Key, typename Value, nvbench::int32_t CGSize, nvbench::int32_t BufferSize>
std::enable_if_t<(sizeof(Key) == sizeof(Value)), void> nvbench_retrieve(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<CGSize>, nvbench::enum_type<BufferSize>>)
{
  std::size_t const num_keys     = state.int64("NumInputs");
  auto const occupancy           = state.float64("Occupancy");
  std::size_t const size         = num_keys / occupancy;
  std::size_t const multiplicity = state.int64("Multiplicity");

  state.add_element_count(num_keys, "NumKeys");
  state.add_global_memory_writes<Key>(num_keys * 2);

  std::vector<Key> h_keys(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_multikeys<Key>(h_keys.begin(), h_keys.end(), multiplicity);
  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
  }

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);

  cuco::static_multimap<Key,
                        Value,
                        cuda::thread_scope_device,
                        cuco::cuda_allocator<char>,
                        cuco::double_hashing<CGSize,
                                             cuco::detail::MurmurHash3_32<Key>,
                                             cuco::detail::MurmurHash3_32<Key>>>
    map{size, cuco::sentinel::empty_key<Key>{-1}, cuco::sentinel::empty_value<Value>{-1}};
  map.insert(d_pairs.begin(), d_pairs.end());

  auto const output_size = map.count_outer(d_keys.begin(), d_keys.end());
  thrust::device_vector<cuco::pair_type<Key, Value>> d_results(output_size);

  state.exec(nvbench::exec_tag::sync, [&](nvbench::launch& launch) {
    map.retrieve_outer(d_keys.begin(), d_keys.end(), d_results.data().get(), launch.stream());
  });
}

template <typename Key, typename Value, nvbench::int32_t CGSize, nvbench::int32_t BufferSize>
std::enable_if_t<(sizeof(Key) != sizeof(Value)), void> nvbench_retrieve(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<CGSize>, nvbench::enum_type<BufferSize>>)
{
  state.skip("Key should be the same type as Value.");
}

using key_type    = nvbench::type_list<nvbench::int32_t, nvbench::int64_t>;
using value_type  = nvbench::type_list<nvbench::int32_t, nvbench::int64_t>;
using cg_size     = nvbench::enum_type_list<1, 2, 4, 8, 16, 32>;
using buffer_size = nvbench::enum_type_list<1, 2, 4, 8, 16>;

NVBENCH_BENCH_TYPES(nvbench_retrieve,
                    NVBENCH_TYPE_AXES(key_type, value_type, cg_size, nvbench::enum_type_list<2>))
  .set_type_axes_names({"Key", "Value", "CGSize", "BufferSize"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.4})
  .add_int64_power_of_two_axis("Multiplicity", nvbench::range(0, 8, 1));

NVBENCH_BENCH_TYPES(
  nvbench_retrieve,
  NVBENCH_TYPE_AXES(key_type, value_type, nvbench::enum_type_list<8>, buffer_size))
  .set_type_axes_names({"Key", "Value", "CGSize", "BufferSize"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.4})
  .add_int64_power_of_two_axis("Multiplicity", nvbench::range(0, 8, 1));
